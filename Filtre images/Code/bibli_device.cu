
#include <hip/hip_runtime.h>
//##############################################################################################
//################################### FONCTION DEVICE ##########################################
//##############################################################################################


__global__ void Filtre(unsigned int *d_img, unsigned width, unsigned height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int n_x = gridDim.x*blockDim.x; // nombre de threads par ligne
	int n_y = gridDim.y*blockDim.y; // nombres de threads par colonne

	int nb_sleep_thread_x = n_x-width; // nombres de threads inatif par ligne
  	int nb_sleep_thread_y = n_y-height; // nombre de threads inactif par colonne

	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){

			int ida = ((y * width) + x) * 3;
			d_img[ida] = 255;

		}
	}
}

__global__ void Sym_horizontale(unsigned int *d_img, unsigned int *d_tmp, unsigned width, unsigned height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int n_x = gridDim.x*blockDim.x; // nombre de threads par ligne
	int n_y = gridDim.y*blockDim.y; // nombres de threads par colonne

	int nb_sleep_thread_x = n_x-width; // nombres de threads inatif par ligne
  	int nb_sleep_thread_y = n_y-height; // nombre de threads inactif par colonne

	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){

				int ida = ((y * width) + x) * 3;
				int idb = (width * height - y*width - width + x) * 3;
				d_img[ida+0] = d_tmp[idb+0];
				d_img[ida+1] = d_tmp[idb+1];
				d_img[ida+2] = d_tmp[idb+2];
		}
	}
}

__global__ void Floutage(unsigned int *d_img, unsigned int *d_tmp, unsigned width, unsigned height, int p) // FLOU
{
	int n_x = gridDim.x*blockDim.x; // nombre de threads par ligne
	int n_y = gridDim.y*blockDim.y; // nombres de threads par colonne

	int nb_sleep_thread_x = n_x-width; // nombres de threads inatif par ligne
	int nb_sleep_thread_y = n_y-height; // nombre de threads inactif par colonne

	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){
			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			int idx = ((y * width) + x) * 3;
		  	int idg = ((y*width) + x-1) * 3; //idg = id pixel à gauche
		  	int idd = ((y*width) + x+1) * 3; //idd = id pixel à droite
		  	int idb = (((y-1)*width) + x) * 3; //idb = id pixel en bas
		  	int idh = (((y+1)*width) + x) * 3; //idh = id pixel du dessus
	
		  	int cond_BAS = (blockIdx.y==0 && threadIdx.y == 0); // bas
		  	int cond_HAUT = (blockIdx.y==gridDim.y-1 && threadIdx.y == blockDim.y-nb_sleep_thread_y-1); // haut
		  	int cond_GAUCHE = (blockIdx.x==0 && threadIdx.x == 0); // gauche
		  	int cond_DROITE = (blockIdx.x==gridDim.x-1 && threadIdx.x == blockDim.x-nb_sleep_thread_x-1); // droite
	
		  	int cond_coin_1 = cond_BAS*cond_GAUCHE; // bas gauche
		  	int cond_coin_2 = cond_BAS*cond_DROITE; // bas droite
		  	int cond_coin_3 = cond_HAUT*cond_GAUCHE; // haut gauche
		  	int cond_coin_4 = cond_HAUT*cond_DROITE; // haut droite

				if(cond_BAS) // bas
				{
			    if(cond_coin_1){ // gauche
						for(int k=0;k<3;k++){
							if(p%2==0){
								d_img[idx+k] = (d_img[idx+k]+d_img[idd+k]+d_img[idh+k])/3;
							}
							else{
								d_img[idx+k] = ceil((d_img[idx+k]+d_img[idd+k]+d_img[idh+k])/3.);
							}
						}
			    }

			    else if(cond_coin_2){ // droite
						for(int k=0;k<3;k++){
							if(p%2==0){
								d_img[idx+k] = (d_img[idx+k]+d_img[idg+k]+d_img[idh+k])/3;
							}
							else{
								d_img[idx+k] = ceil((d_img[idx+k]+d_img[idg+k]+d_img[idh+k])/3.);
							}
						}
			    }

			    else{
						for(int k=0;k<3;k++){
							if(p%2==0){
								d_img[idx+k] = (d_img[idx+k]+d_img[idg+k]+d_img[idd+k]+d_img[idh+k])/4;
							}
							else{
								d_img[idx+k] = ceil((d_img[idx+k]+d_img[idg+k]+d_img[idd+k]+d_img[idh+k])/4.);
							}
						}
			    }
			  }

				else if(cond_HAUT) // haut
				{
			    if(cond_coin_3){ // gauche
						for(int k=0;k<3;k++){
							if(p%2==0){
								d_img[idx+k] = (d_img[idx+k]+d_img[idd+k]+d_img[idb+k])/3;
							}
							else{
								d_img[idx+k] = ceil((d_img[idx+k]+d_img[idd+k]+d_img[idb+k])/3.);
							}
						}
			    }

			    else if(cond_coin_4){ // droite
						for(int k=0;k<3;k++){
							if(p%2==0){
								d_img[idx+k] = (d_img[idx+k]+d_img[idg+k]+d_img[idb+k])/3;
							}
							else{
								d_img[idx+k] = ceil((d_img[idx+k]+d_img[idg+k]+d_img[idb+k])/3.);
							}
						}
			    }
			    else{
						for(int k=0;k<3;k++){
							if(p%2==0){
								d_img[idx+k] = (d_img[idx+k]+d_img[idg+k]+d_img[idd+k]+d_img[idb+k])/4;
							}
							else{
								d_img[idx+k] = ceil((d_img[idx+k]+d_img[idg+k]+d_img[idd+k]+d_img[idb+k])/4.);
							}
						}
			    }
			  }

				else if(cond_GAUCHE) // gauche
				{
					for(int k=0;k<3;k++){
						if(p%2==0){
							d_img[idx+k] = (d_img[idx+k]+d_img[idd+k]+d_img[idb+k]+d_img[idh+k])/4;
						}
						else{
							d_img[idx+k] = ceil((d_img[idx+k]+d_img[idd+k]+d_img[idb+k]+d_img[idh+k])/4.);
						}
					}
			  }

				else if(cond_DROITE) // droite
				{
					for(int k=0;k<3;k++){
						if(p%2==0){
							d_img[idx+k] = (d_img[idx+k]+d_img[idg+k]+d_img[idb+k]+d_img[idh+k])/4;
						}
						else{
							d_img[idx+k] = ceil((d_img[idx+k]+d_img[idg+k]+d_img[idb+k]+d_img[idh+k])/4.);
						}
					}
			  }

				else
			  {
					for(int k=0;k<3;k++){
						if(p%2==0){
							d_img[idx+k] = (d_img[idx+k]+d_img[idg+k]+d_img[idd+k]+d_img[idb+k]+d_img[idh+k])/5;
						}
						else{
							d_img[idx+k] = ceil((d_img[idx+k]+d_img[idg+k]+d_img[idd+k]+d_img[idb+k]+d_img[idh+k])/5.);
						}
					}
				}
		}
	}
}

__global__ void Niveau_Gris(unsigned int *d_tmp, unsigned width, unsigned height) // NIVEAU DE GRIS
{
	int n_x = gridDim.x*blockDim.x; // nombre de threads par ligne
	int n_y = gridDim.y*blockDim.y; // nombres de threads par colonne

	int nb_sleep_thread_x = n_x-width; // nombres de threads inatif par ligne
	int nb_sleep_thread_y = n_y-height; // nombre de threads inactif par colonne

	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){
			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			int idx = ((y * width) + x) * 3;

			d_tmp[idx+0] = 0.299*d_tmp[idx+0]+0.587*d_tmp[idx+1]+0.114*d_tmp[idx+2];
			d_tmp[idx+1] = d_tmp[idx+0];
			d_tmp[idx+2] = d_tmp[idx+0];
		}
	}
}

__global__ void Contour_Sobel(unsigned int *d_img, unsigned int *d_tmp, unsigned width,unsigned height, // SOBEL CONTOUR
											 				int *d_filtre)
{
	int n_x = gridDim.x*blockDim.x; // nombre de threads par ligne
	int n_y = gridDim.y*blockDim.y; // nombres de threads par colonne

	int nb_sleep_thread_x = n_x-width; // nombres de threads inatif par ligne
	int nb_sleep_thread_y = n_y-height; // nombre de threads inactif par colonne

	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){
			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			int idx = ((y * width) + x) * 3;

			int idhg = (((y+1)*width) + x-1) * 3;
			int idh = (((y+1)*width) + x) * 3; //idh = id pixel du dessus
			int idhd = (((y+1)*width) + x+1) * 3;
			int idg = ((y*width) + x-1) * 3; //idg = id pixel à gauche
			int idd = ((y*width) + x+1) * 3; //idd = id pixel à droite
			int idbg = (((y-1)*width) + x-1) * 3;
			int idb = (((y-1)*width) + x) * 3; //idb = id pixel en bas
			int idbd = (((y-1)*width) + x+1) * 3;

			int cond_BAS = (blockIdx.y==0 && threadIdx.y==0); // bas
			int cond_HAUT = (blockIdx.y==gridDim.y-1 && threadIdx.y==blockDim.y-nb_sleep_thread_y-1); // haut
			int cond_GAUCHE = (blockIdx.x==0 && threadIdx.x==0); // gauche
			int cond_DROITE = (blockIdx.x==gridDim.x-1 && threadIdx.x==blockDim.x-nb_sleep_thread_x-1); // droite

			int cond_coin_1 = cond_BAS*cond_GAUCHE; // bas gauche
			int cond_coin_2 = cond_BAS*cond_DROITE; // bas droite
			int cond_coin_3 = cond_HAUT*cond_GAUCHE; // haut gauche
			int cond_coin_4 = cond_HAUT*cond_DROITE; // haut droite

			//on applique maintenant les filtres de Sobel
			int d_G[2] = {0,0};
			int id[9] = {idhg,idh,idhd,idg,idx,idd,idbg,idb,idbd};

			if(cond_BAS) // BAS
			{
		    if(cond_coin_1) // gauche
				{
					for(int k=0; k<9;k++){
						d_G[0] += d_tmp[id[k]+3*width+3]*d_filtre[k];
						d_G[1] += d_tmp[id[k]+3*width+3]*d_filtre[k+9];
					}
		    }

		    else if(cond_coin_2)//droite
				{
					for(int k=0; k<9;k++){
						d_G[0] += d_tmp[id[k]+3*width-3]*d_filtre[k];
						d_G[1] += d_tmp[id[k]+3*width-3]*d_filtre[k+9];
					}
		    }

		    else
				{
					for(int k=0; k<9;k++){
						d_G[0] += d_tmp[id[k]+3*width]*d_filtre[k];
						d_G[1] += d_tmp[id[k]+3*width]*d_filtre[k+9];
					}
		    }
		  }

			else if(cond_HAUT) // HAUT
			{
		    if(cond_coin_3) // gauche
				{
					for(int k=0; k<9;k++){
						d_G[0] += d_tmp[id[k]-3*width+3]*d_filtre[k];
						d_G[1] += d_tmp[id[k]-3*width+3]*d_filtre[k+9];
					}
		    }

		    else if(cond_coin_4) //droite
				{
					for(int k=0; k<9;k++){
						d_G[0] += d_tmp[id[k]-3*width-3]*d_filtre[k];
						d_G[1] += d_tmp[id[k]-3*width-3]*d_filtre[k+9];
					}
		    }
		    else
				{
					for(int k=0; k<9;k++){
						d_G[0] += d_tmp[id[k]-3*width]*d_filtre[k];
						d_G[1] += d_tmp[id[k]-3*width]*d_filtre[k+9];
					}
		    }
		  }

			else if(cond_GAUCHE) // GAUCHE
			{
				for(int k=0; k<9;k++){
					d_G[0] += d_tmp[id[k]+3]*d_filtre[k];
					d_G[1] += d_tmp[id[k]+3]*d_filtre[k+9];
				}
		  }

			else if(cond_DROITE) // DROITE
			{
				for(int k=0; k<9;k++){
					d_G[0] += d_tmp[id[k]-3]*d_filtre[k];
					d_G[1] += d_tmp[id[k]-3]*d_filtre[k+9];
				}
		  }

			else // CAS GENERAL
		  {
				for(int k=0; k<9;k++){
					d_G[0] += d_tmp[id[k]]*d_filtre[k];
					d_G[1] += d_tmp[id[k]]*d_filtre[k+9];
				}
			}
			d_img[idx] = (int)pow(d_G[0]*d_G[0] + d_G[1]*d_G[1],0.5);
			d_img[idx+1] = d_img[idx];
			d_img[idx+2] = d_img[idx];
		}
	}
}

__global__ void Pop_art(unsigned int *d_img, unsigned int *d_tmp, unsigned width, unsigned height)
{
	int n_x = gridDim.x*blockDim.x; // nombre de threads par ligne
	int n_y = gridDim.y*blockDim.y; // nombres de threads par colonne

	int nb_sleep_thread_x = n_x-width; // nombres de threads inatif par ligne
	int nb_sleep_thread_y = n_y-height; // nombre de threads inactif par colonne

	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){
			int x = blockIdx.x * blockDim.x + threadIdx.x - blockIdx.y*nb_sleep_thread_x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			int ida  = ((y * width) + x) * 3;

			if((x <= width/2) && (y < height/2)){ // BAS GAUCHE
				d_img[ida+0] = 255;
			}
			else if((x > width/2) && (y < height/2)){ // BAS DROITE
				d_img[ida+1] = 255;
			}
			else if((x > width/2) && (y >= height/2)){ //HAUT DROITE
				d_img[ida+2] = 255;
			}
		}
	}
}

__global__ void Pop_art_stream(unsigned int *d_img, unsigned width, unsigned height,int taille,int p)
{
	int n_x = gridDim.x*blockDim.x; // nombre de threads par ligne
	int n_y = gridDim.y*blockDim.y; // nombres de threads par colonne

	int nb_sleep_thread_x = n_x-width; // nombres de threads inatif par ligne
	int nb_sleep_thread_y = n_y-height; // nombre de threads inactif par colonne

	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){
			int x = blockIdx.x * blockDim.x + threadIdx.x - blockIdx.y*nb_sleep_thread_x;
			int y = p*n_y + blockIdx.y * blockDim.y + threadIdx.y;

			int ida  = ((y * width) + x) * 3;

			if((x <= width/2) && (y < height/2)){
				d_img[ida+0] = 255;
			}
			else if((x > width/2) && (y < height/2)){
				d_img[ida+1] = 255;
			}
			else if((x > width/2) && (y >= height/2)){
				d_img[ida+2] = 255;
			}
		}
	}
}


//##############################################################################################
//################################### EN PLUS ##################################################
//##############################################################################################




__global__ void AUTRE_Pop_art(unsigned int *d_img, unsigned int *d_tmp, unsigned width, unsigned height)
{
	int n_x = gridDim.x*blockDim.x; // nombre de threads par ligne
	int n_y = gridDim.y*blockDim.y; // nombres de threads par colonne

	int nb_sleep_thread_x = n_x-width; // nombres de threads inatif par ligne
	int nb_sleep_thread_y = n_y-height; // nombre de threads inactif par colonne

	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){
			int x = blockIdx.x * blockDim.x + threadIdx.x - blockIdx.y*nb_sleep_thread_x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			int ida  = ((y * width) + x) * 3;

			if((x <= width/2) && (y < height/2)){ // BAS GAUCHE
				int idb = (((y+height/2) * width) + x) * 3;
				d_img[ida+0] = 255;
				d_img[ida+1] = d_tmp[idb+1];
				d_img[ida+2] = d_tmp[idb+2];
			}
			else if((x > width/2) && (y < height/2)){ // BAS DROITE
				int idb = (((y+height/2) * width) + x - width/2) * 3;
				d_img[ida+0] = d_tmp[idb+0];
				d_img[ida+1] = 255;
				d_img[ida+2] = d_tmp[idb+2];
			}
			else if((x > width/2) && (y >= height/2)){ //HAUT DROITE
				int idb = ((y * width) + x - width/2) * 3;
				d_img[ida+0] = d_tmp[idb+0];
				d_img[ida+1] = d_tmp[idb+1];
				d_img[ida+2] = 255;
			}
		}
	}
}

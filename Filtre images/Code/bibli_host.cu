//##########################################################################################
//################################### HOST #################################################
//##########################################################################################


double get_time() {
  struct timeval tv;
  gettimeofday(&tv, (void *)0);
  return (double) tv.tv_sec + tv.tv_usec*1e-6;
}

__host__ int error_msg(int argc,int argv1, int argv2, int argv3){
  int q = (argv3 == 0) + (argv3 == 6) + (argv3 == 7) + (argv3 == 8) + (argv3 == 9) + (argv3 == 10) + (argv3 == 12) + (argv3 == 14);
	if(argc != 4 || !q){
		printf("Usage : le programme prend 3 arguments.\n"
					"argv[1] = le nombre de threads par bloc selon x.\n"
					"argv[2] = le nombre de threads par bloc selon y.\n"
					"argv[3] = le numéro de la question qu'on souhaite réaliser (6,7,8,9,10,12,14).\n\n"
          "Exemple: ./modif_img.exe 32 20 6 \n\n");
    return 1;
	}
  if(argv1*argv2 > 1024){
    printf("\nERROR : Nombre de threads par bloc possible dépassé, argv[2]*argv[3] doit etre inférieur à 1024. \n\n");
    return 1;
  }
  return 0;
}

__host__ void REORDER_IMG(unsigned int *img,int  height, int width,unsigned pitch,FIBITMAP* bitmap){
	BYTE *bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = bits;
    for ( int x =0; x<width; x++)
    {
      int idx = ((y * width) + x) * 3;
      img[idx + 0] = pixel[FI_RGBA_RED];
      img[idx + 1] = pixel[FI_RGBA_GREEN];
      img[idx + 2] = pixel[FI_RGBA_BLUE];
      pixel += 3;
    }
    bits += pitch;
  }
}

__host__ void SAVE_IMG(unsigned int *img,int  height, int width,const char *PathDest,const char *PathSave,unsigned pitch,FIBITMAP* bitmap){
	BYTE* bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = (BYTE*)bits;
    for ( int x =0; x<width; x++)
    {
      RGBQUAD newcolor;

      int idx = ((y * width) + x) * 3;
      newcolor.rgbRed = img[idx + 0];
      newcolor.rgbGreen = img[idx + 1];
      newcolor.rgbBlue = img[idx + 2];

      if(!FreeImage_SetPixelColor(bitmap, x, y, &newcolor))
      { fprintf(stderr, "(%d, %d) Fail...\n", x, y); }

      pixel+=3;
    }
    // next line
    bits += pitch;
  }

  if( FreeImage_Save (FIF_PNG, bitmap , PathSave , 0 ))
    printf("Image successfully saved ! ");
  FreeImage_DeInitialise(); //Cleanup !
	printf("bip bip bip\n");
}


__host__ void free_all(unsigned int* d_img,unsigned int *img, unsigned int *d_tmp,int *d_filtre,int* filtre){
	hipFree(d_img);
  hipFree(d_tmp);
	hipFree(d_filtre);

	hipFree(img);
	hipFree(filtre);
}

__host__ void affichage(int block_x, int block_y, int grid_x, int grid_y, int height, int width){

	int nb_sleep_thread_x = grid_x*block_x-width; // nombres de threads inatif par ligne
	int nb_sleep_thread_y = grid_y*block_y-height;
	printf("\n");
	printf("##############   blockDim.x = %d\n",block_x);
	printf("##############   gridDim.x = %d\n",grid_x);
	printf("##############   blockDim.y = %d\n",block_y);
	printf("##############   gridDim.y = %d\n",grid_y);
	printf("##############   nombre de threads inactif par ligne = %d\n",nb_sleep_thread_x);
	printf("##############   nombre de threads inactif par colonne = %d\n",nb_sleep_thread_y);
	printf("\n");
}


__host__ void filtre_SOBEL(int *filtre){
	//############# FILTRE NORMAL ########################
	//déclaration et initialisation filtre horizontal
	filtre[0] = filtre[6] = -1;
	filtre[1] = filtre[4] = filtre[7] = 0;
	filtre[2] = filtre[8] = 1;
	filtre[3] = -2;
	filtre[5] = 2;

	//déclaration et initialisation filtre vertical
	filtre[9] = filtre[11] = -1;
	filtre[12] = filtre[13] = filtre[14] = 0;
	filtre[15] = filtre[17] = 1;
	filtre[10] = -2;
	filtre[16] = 2;
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <sys/time.h>
#include "FreeImage.h"
#include "bibli_host.cu"
#include "bibli_device.cu"


#define WIDTH 1920
#define HEIGHT 1024
#define BPP 24 // Since we're outputting three 8 bit RGB values

typedef float              f32;
typedef double             f64;
typedef unsigned long long u64;

using namespace std;

int main (int argc , char** argv)
{
  double start,stop,cpu_time_used;

  int err = 0;
  err = error_msg(argc,atoi(argv[1]),atoi(argv[2]),atoi(argv[3])); // gestion des erreur des arguments d'entrée
  if(err==1){
    return 0;
  }

  FreeImage_Initialise();
  const char *PathName = "frame27.jpg";
  const char *PathDest,*PathSave;

  // load and decode a regular file
  FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);
  FIBITMAP* bitmap = FreeImage_Load(FIF_JPEG, PathName, 0);

  if(! bitmap )
    exit( 1 ); //WTF?! We can't even allocate images ? Die !

  unsigned width  = FreeImage_GetWidth(bitmap);
  unsigned height = FreeImage_GetHeight(bitmap);
  unsigned pitch  = FreeImage_GetPitch(bitmap);

  fprintf(stderr, "Processing Image of size %d x %d\n", width, height);

	int sz_in_bytes = sizeof(unsigned int) * 3 * width * height; //nb de valeurs pour toute image

  //déclaration host
	unsigned int *img;	//image de départ
  int *filtre;
  //allocation host
  img = (unsigned int*) malloc(sz_in_bytes);
  filtre = (int*) malloc(sizeof(int)*18);
  filtre_SOBEL(filtre);

	//déclaration device
	unsigned int *d_img;
	unsigned int *d_tmp;
  unsigned int *d_tmp_N;
  int *d_filtre;
	//allocation device
	hipMalloc((void**)&d_img, sz_in_bytes);
  hipMalloc((void**)&d_tmp, sz_in_bytes);
  hipMalloc((void**)&d_tmp_N, sz_in_bytes);
  hipMalloc((void**)&d_filtre,sizeof(int)*18);

  REORDER_IMG(img,height,width,pitch,bitmap);

  hipMemcpy(d_img, img, sz_in_bytes,hipMemcpyHostToDevice);
  hipMemcpy(d_tmp, img, sz_in_bytes,hipMemcpyHostToDevice);
  hipMemcpy(d_filtre, filtre, sizeof(int)*18,hipMemcpyHostToDevice);

  int block_x = atoi(argv[1]);
  int grid_x = ceil((float)width/(float)block_x);
  int block_y = atoi(argv[2]);
  int grid_y = ceil((float)height/(float)block_y);

  affichage(block_x,block_y,grid_x,grid_y,height,width);

  dim3 dimBlock(block_x,block_y,1);
  dim3 dimGrid(grid_x,grid_y,1);

  start = get_time();

  //############################## TEST ##############################
	if(atoi(argv[3])==0){

		for(int k=0;k<3*100000;k=k+3){
      img[k+0] = 255;
      img[k+1] = 0;
      img[k+2] = 0;
    }

		PathDest = "test.png";
		PathSave = "ImageRapport/test.png";
		printf("Execute: Un test\n\n");

		SAVE_IMG(img,height,width,PathDest,PathSave,pitch,bitmap);
	}

	//############################## Saturation rouge ##############################
	if(atoi(argv[3])==6){

		Filtre<<<dimGrid, dimBlock>>>(d_img, width, height);

		hipMemcpy(img, d_img, sz_in_bytes,hipMemcpyDeviceToHost);

		PathDest = "Saturation_rouge.png";
		PathSave = "ImageRapport/Saturation_rouge.png";
		printf("Execute: Saturation rouge\n\n");

		SAVE_IMG(img,height,width,PathDest,PathSave,pitch,bitmap);
	}

	//############################## Symétrie horizontale ##############################
	if(atoi(argv[3])==7){

		Sym_horizontale<<<dimGrid, dimBlock>>>(d_img, d_tmp, width, height);

		hipMemcpy(img, d_img, sz_in_bytes,hipMemcpyDeviceToHost);

		PathDest = "Symétrie_horizontale.png";
		PathSave = "ImageRapport/Symétrie_horizontale.png";
		printf("Execute: Symétrie horizontale\n\n");

		SAVE_IMG(img,height,width,PathDest,PathSave,pitch,bitmap);
	}

	//############################## Floutage ##############################
  if(atoi(argv[3])==8){

    int nb_flou = 100; // Nombre de floutage consécutifs
    for(int p = 0; p < nb_flou; p++){
      Floutage<<<dimGrid, dimBlock>>>(d_img, d_tmp, width, height,p);
    }

    hipMemcpy(img, d_img, sz_in_bytes,hipMemcpyDeviceToHost);

    PathDest = "Floutage.png";
    PathSave = "ImageRapport/Floutage.png";
    printf("Execute: Floutage\n\n");

    SAVE_IMG(img,height,width,PathDest,PathSave,pitch,bitmap);
  }

  //############################## Niveau de gris ##############################
  if(atoi(argv[3])==9){

    Niveau_Gris<<<dimGrid, dimBlock>>>(d_img, width, height);

    hipMemcpy(img, d_img, sz_in_bytes,hipMemcpyDeviceToHost);

    PathDest = "Niveau_gris.png";
    PathSave = "ImageRapport/Niveau_gris.png";
    printf("Execute: Niveau de Gris\n\n");

    SAVE_IMG(img,height,width,PathDest,PathSave,pitch,bitmap);
  }

  //############################## Contour Sobel ##############################
  if(atoi(argv[3])==10){

    Niveau_Gris<<<dimGrid, dimBlock>>>(d_tmp, width, height);
  	Contour_Sobel<<<dimGrid, dimBlock>>>(d_img, d_tmp, width, height, d_filtre);

    hipMemcpy(img, d_img, sz_in_bytes,hipMemcpyDeviceToHost);

    PathDest = "Contour_Sobel.png";
    PathSave = "ImageRapport/Contour_Sobel.png";
    printf("Execute: Contour Sobel\n\n");

    SAVE_IMG(img,height,width,PathDest,PathSave,pitch,bitmap);
  }

  //############################## Pop_art ##############################
  if(atoi(argv[3])==12){

    Pop_art<<<dimGrid, dimBlock>>>(d_img, d_tmp, width, height);

    hipMemcpy(img, d_img, sz_in_bytes,hipMemcpyDeviceToHost);

    PathDest = "Pop_art.png";
    PathSave = "ImageRapport/Pop_art.png";
    printf("Execute: Pop art\n\n");

    SAVE_IMG(img,height,width,PathDest,PathSave,pitch,bitmap);
  }

  //############################## Pop art stream ##############################
  if(atoi(argv[3])==14){
    hipStream_t stream[4];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);
    hipStreamCreate(&stream[2]);
    hipStreamCreate(&stream[3]);

    int taille = 3 * width * height/4;

    dim3 dimBlock(block_x,block_y,1);
    dim3 dimGrid(grid_x,grid_y/4,1);

    hipMemcpyAsync(d_img, img, sz_in_bytes,hipMemcpyHostToDevice,stream[0]);

    Pop_art_stream<<<dimGrid, dimBlock, 0, stream[0]>>>(d_img, width, height, taille, 0);
    Pop_art_stream<<<dimGrid, dimBlock, 0, stream[1]>>>(d_img, width, height, taille, 1);
    Pop_art_stream<<<dimGrid, dimBlock, 0, stream[2]>>>(d_img, width, height, taille, 2);
    Pop_art_stream<<<dimGrid, dimBlock, 0, stream[3]>>>(d_img, width, height, taille, 3);

    hipMemcpyAsync(img, d_img, sz_in_bytes/4,hipMemcpyDeviceToHost,stream[0]);
    hipMemcpyAsync(img+taille, d_img+taille, sz_in_bytes/4,hipMemcpyDeviceToHost,stream[1]);
    hipMemcpyAsync(img+2*taille, d_img+2*taille, sz_in_bytes/4,hipMemcpyDeviceToHost,stream[2]);
    hipMemcpyAsync(img+3*taille, d_img+3*taille, sz_in_bytes/4,hipMemcpyDeviceToHost,stream[3]);

    PathDest = "Pop_art_STREAM.png";
    PathSave = "ImageRapport/Pop_art_STREAM.png";
    printf("Execute: Pop art STREAM\n\n");

    SAVE_IMG(img,height,width,PathDest,PathSave,pitch,bitmap);
  }

  stop = get_time();
  cpu_time_used = stop-start;
  printf("\ntemps %f\n",cpu_time_used);

  free_all(d_img,img,d_tmp,d_filtre,filtre);
}

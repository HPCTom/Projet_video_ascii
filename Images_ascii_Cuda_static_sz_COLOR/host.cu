#include "hip/hip_runtime.h"
//##############################################################################################
//################################### FONCTION HOST ############################################
//##############################################################################################

__host__ void description_parametre(int arg0, int arg1, int arg2, int *max_it){
  char ascii[10];
  char fond[10];
  if(arg1 == 0){
    sprintf(ascii, "%s", "noir");
  }
  else if (arg1 == 255){
    sprintf(ascii, "%s", "blanc");
  }
  else{
    sprintf(ascii, "%s", "couleur");
  }

  if(arg2 == 0){
    sprintf(fond, "%s", "noir");
  }
  else if (arg2 == 255){
    sprintf(fond, "%s", "blanc");
  }
  else{
    sprintf(fond,"%s", "couleur");
  }

  char nbr_img[100] = {0}; 

  FILE * f_img = popen("find images -type f | wc -l","r");
  fgets(nbr_img, 100, f_img); // calcul le nombre d'image à transformer
  pclose(f_img);
  *max_it = atoi(nbr_img);

  printf("Traduction ascii à %d%% avec ascii %s et fond %s pour %d images \n",arg0,ascii,fond,*max_it);

}

void list_dir(char *path,int *cpt,char **path_dir,int mode)
{ 

  DIR * d = opendir(path); // open the path
  if(d==NULL) return; // if was not able, return
  struct dirent * dir; // for the directory entries

  while ((dir = readdir(d)) != NULL) // if we were able to read somehting from the directory
    {

      if(dir -> d_type == DT_DIR && strcmp(dir->d_name,".")!=0 && strcmp(dir->d_name,"..")!=0 && mode == 0 ) // if it is a directory
      {
        *cpt = *cpt+1;
        char d_path[257]; // here I am using sprintf which is safer than strcat
        sprintf(d_path, "%s/%s", path, dir->d_name);
        list_dir(d_path,cpt,path_dir,0);
      }
      if(dir -> d_type == DT_DIR && strcmp(dir->d_name,".")!=0 && strcmp(dir->d_name,"..")!=0 && mode == 1 ) // if it is a directory
      {
        char d_path[257]; // here I am using sprintf which is safer than strcat
        sprintf(d_path, "%s/%s", path, dir->d_name);
        sprintf(path_dir[*cpt], "%s", d_path);
        char ascii_path_dir[255] = "mkdir images_ascii";
        strcat(ascii_path_dir, &path_dir[*cpt][6]);

        //printf("%s \n",ascii_path_dir);
        system(ascii_path_dir);

        *cpt = *cpt+1;
        list_dir(d_path,cpt,path_dir,1);
        
      }
    }
    closedir(d); // finally close the directory
}

void show_dir_content(char *path, int nb_img, char **path_img,int *ite)
{ 

  DIR * d = opendir(path); // open the path
  if(d==NULL) return; // if was not able, return
  struct dirent * dir; // for the directory entries

  while ((dir = readdir(d)) != NULL) // if we were able to read somehting from the directory
    {
      if(dir-> d_type != DT_DIR)
      { // if the type is not directory just print it with blue color
        sprintf(path_img[*ite], "%s/%s", path,dir->d_name);
        *ite = *ite+1;
      }
      else
      if(dir -> d_type == DT_DIR && strcmp(dir->d_name,".")!=0 && strcmp(dir->d_name,"..")!=0 ) // if it is a directory
      {
        char d_path[257]; // here I am using sprintf which is safer than strcat
        sprintf(d_path, "%s/%s", path, dir->d_name);
        show_dir_content(d_path,nb_img,path_img,ite); // recall with the new path
      }
    }
    closedir(d); // finally close the directory
}

__host__ void find_path_img(char **path_img, int *max_it){

  char nbr_img[100] = {0}; 

  FILE * f_img = popen("find images -type f | wc -l","r");
  fgets(nbr_img, 100, f_img); // calcul le nombre d'image à transformer
  pclose(f_img);

  *max_it = atoi(nbr_img);
  //printf("max_it = %d \n",*max_it);

  int ite = 0;
  show_dir_content("images",atoi(nbr_img),path_img,&ite);
}

__host__ void error_msg(int msg_case,const char msg){
  switch(msg_case){
    case 0:
      printf("\nUsage : le programme prend au moins 3 arguments.\n"
      "argv[1] = pourcentage de résolution de l'image entre 0 et 100 (100 %% indique qu'il y aura autant d'ascii en largeur que de pixels).\n"
      "argv[2] = nombre d'ascii différents utilisés pour générer l'image.\n"
      "argv[3] = nom de la video.\n"
      "Exemple: ./modif_img 50 10 my_video.mp4 \n\n");
      break;

    case 1:
      printf("ERROR argument invalide : %c \n",msg);
      break;
  }
}

//#### Gestion des erreurs dans les parametres d'entré #####
__host__ void ARG_ERROR(int argc,char argv1,char argv2){
  int nb_max_arg = 5;

  const char *default_msg[4] = {"0", "orange", "yellow", "blue"};

	if(argc > nb_max_arg || argc < 4){
		error_msg(0,*default_msg[0]);
    exit (EXIT_FAILURE);
	}

  else if(atoi(&argv1) <= 0 || atoi(&argv1) > 100){
    error_msg(0,*default_msg[0]);
    error_msg(1,argv1);
    exit (EXIT_FAILURE);
  }

  else if(atoi(&argv2) <= 0 || atoi(&argv2) > 10){
    error_msg(0,*default_msg[0]);
    error_msg(1,argv2);
    exit (EXIT_FAILURE);
  }

  else{
    printf("Pas de problèmes dans les arguments \n");
  }




}

//#### Declaration des parametres
__host__ void declaration_1(FIBITMAP *bitmap,unsigned *width,unsigned *height,unsigned *pitch){
    *width  = FreeImage_GetWidth(bitmap);
    *height = FreeImage_GetHeight(bitmap);
    *pitch  = FreeImage_GetPitch(bitmap);
    // printf("pitch = %d \n",*pitch);
}

__host__ void declaration_2(unsigned int width, unsigned int height,unsigned int *blockDim_x,unsigned int *blockDim_y,unsigned int *gridDim_x,unsigned int *gridDim_y,
                            unsigned int *blockDim_x_ascii,unsigned int *blockDim_y_ascii,unsigned int *gridDim_x_ascii,unsigned int *gridDim_y_ascii,
                            unsigned int *nb_sleep_thread_x,unsigned int *nb_sleep_thread_y,unsigned int *nb_sleep_thread_x_ascii,unsigned int *nb_sleep_thread_y_ascii,
                            float poucrentage_image){                                                       

  // #### GRILLE GENERALE ####
  *blockDim_x = 16; // ok 
  *blockDim_y = 16; // ok
  *gridDim_x = ceil((float)width/(float)(*blockDim_x)); // ok 
  *gridDim_y = ceil((float)height/(float)(*blockDim_y)); // ok
  *nb_sleep_thread_x = *gridDim_x*(*blockDim_x)-width; // nombres de threads inatif par ligne
  *nb_sleep_thread_y = *gridDim_y*(*blockDim_y)-height; // nombre de threads inactif par colonne

  // #### DIMENSION ASCII FINALE ####
  int nb_ascii_largeur = ceil(((float)width*poucrentage_image/100.));       //nombre d'ascii qu'il y aura en largeur sur l'image finale
  float ratio = (float)width/(float)height*1.5;                             //ratio largeur/hauteur pour calculer la hauteur en ascii de l'image finale (1.5 prend en compte l'écart plus grand entre les lignes que les colonnes dans un fichier texte)
  int nb_ascii_hauteur = ceil(((float)nb_ascii_largeur/ratio));             //nombre d'ascii qu'il y aura en largeur sur l'image finale

  // //### GRILLE SECONDAIRE ###
  int gridDim_x_ascii_all = nb_ascii_largeur; 
	int gridDim_y_ascii_all = nb_ascii_hauteur;

  *blockDim_x_ascii = ceil((float)width/(float)(gridDim_x_ascii_all));
  *blockDim_y_ascii = ceil((float)height/(float)(gridDim_y_ascii_all));

  int nb_sleep_thread_x_ascii_all = gridDim_x_ascii_all*(*blockDim_x_ascii)-width;   // nombres de threads inactif selon x
  int nb_sleep_thread_y_ascii_all = gridDim_y_ascii_all*(*blockDim_y_ascii)-height;  // nombres de threads inactif selon y
  int nb_sleep_block_x_all = (int)((float)nb_sleep_thread_x_ascii_all/(float)(*blockDim_x_ascii)); // nombre de blocks inactifs selon x pour le premier block qui dépasse (arrondie inférieur)
  int nb_sleep_block_y_all = (int)((float)nb_sleep_thread_y_ascii_all/(float)(*blockDim_y_ascii)); // nombre de blocs inactis selon y pour le premier block qui dépasse (arrondie inférieur)

  *gridDim_x_ascii = gridDim_x_ascii_all - nb_sleep_block_x_all; // nouvelle grid en x redimensionnée 
  *gridDim_y_ascii = gridDim_y_ascii_all - nb_sleep_block_y_all; // nouvelle grid en y redimensionnée 

  *nb_sleep_thread_x_ascii = *gridDim_x_ascii*(*blockDim_x_ascii)-width; // nombres de threads inatif par ligne
  *nb_sleep_thread_y_ascii = *gridDim_y_ascii*(*blockDim_y_ascii)-height; // nombre de threads inactif par colonne

}

__host__ void declaration_3(unsigned int blockDim_x_color,unsigned int blockDim_y_color,unsigned int *gridDim_x_color,unsigned int *gridDim_y_color,
                            unsigned int gridDim_x_ascii,unsigned int gridDim_y_ascii, long unsigned int *sz_in_bytes_ascii_color,
                            unsigned int *width_color,unsigned int *height_color){   


  *width_color = blockDim_x_color*gridDim_x_ascii;
  *height_color = blockDim_y_color*gridDim_y_ascii;                          

  *gridDim_x_color = gridDim_x_ascii;
  *gridDim_y_color = gridDim_y_ascii; 

  *sz_in_bytes_ascii_color = 3*blockDim_x_color*blockDim_y_color*gridDim_x_ascii*gridDim_y_ascii*sizeof(unsigned int);

}


void init_barre_chargement(char *barre,int *cpt,float eps,int max_it){
  strcat(barre, " [");
  while(barre[*cpt]!='\0'){
    *cpt = *cpt+1;
  }
  int taille = (int)ceil(100./eps); //nombre de #

  for(int t=0;t<taille;t++){
    strcat(barre, " ");
  }
  strcat(barre, "]");
}

void barre_chargement(char *barre,float p,int k, int max, float eps,int taille){ //entier entre 0 et 100

  int idx = (int)(p/eps); // a quelle intervalle j'appartiens

  if(k==max){
    for(int k=taille; k<taille+idx+1; k++){
      barre[k] = '#';
    }
    printf("\r%s  %.2f%% \n",barre,p);
    fflush(stdout);
  }
  else{
    for(int k=taille; k<taille+idx; k++){
      barre[k] = '#';
    }
    printf("\r%s  %.2f%% ",barre,p);
    fflush(stdout);
  }
}

//#### Permet de calculer le temps #####
__host__ double get_time() {
  struct timeval tv;
  gettimeofday(&tv, (void *)0);
  return (double) tv.tv_sec + tv.tv_usec*1e-6;
}

//#### Ordonne le tableau en RGB? #####
__host__ void REORDER_IMG(unsigned int *img,int  height, int width,unsigned pitch,FIBITMAP* bitmap){
	BYTE *bits = (BYTE*)FreeImage_GetBits(bitmap);
  
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = bits;
    for ( int x =0; x<width; x++)
    {
      int idx = ((y * width) + x) * 3;
      img[idx + 0] = pixel[FI_RGBA_RED];
      img[idx + 1] = pixel[FI_RGBA_GREEN];
      img[idx + 2] = pixel[FI_RGBA_BLUE];
      pixel += 3;
    }
    bits += pitch;
  }
  FreeImage_DeInitialise();
}

//#### Sauvergarde l'image finale #####
__host__ void SAVE_IMG(char *PathName,unsigned char *img,unsigned int height,unsigned int width,unsigned int pitch,FIBITMAP* bitmap,int num){

  FreeImage_Initialise();
  // FreeImage_SetTransparent(bitmap, TRUE);
  // int cpt = 1;
  // float val = (float)num;
  // while (val/10 > 1){
  //   cpt = cpt+1;
  //   val = val/10;
  // }
  // int size_PathDest = 22+cpt;
  // char PathDest[size_PathDest];                            // nom de l'image png de sortie
  // sprintf(PathDest,"images_ascii/frame%d.png", num);
  //strcpy(PathDest,PathName);


  char PathDest[255] = "images_ascii";
  strcat(PathDest, &PathName[6]);

  // for(int kk = 0; kk<sizeof(PathDest)+1;kk++){
  //   printf("%c",PathDest[kk]);
  // }

	// BYTE* bits = (BYTE*)FreeImage_GetBits(bitmap);
  RGBQUAD newcolor;
  // #pragma omp parallel for collapse(2)
  for ( int y =0; y<height; y++)
  {
    // BYTE *pixel = (BYTE*)bits;
    for ( int x =0; x<width; x++)
    {
      // RGBQUAD newcolor;

      int idx = ((y * width) + x) * 3;

      // printf("pitch =  %d, height = %d, width = %d \n",pitch,height,width);
      // if(num == 0){
      //   printf("\nimg[idx + 0] = %d \n",img[idx + 0]);
      //   printf("img[idx + 1] = %d \n",img[idx + 1]);
      //   printf("img[idx + 2] = %d \n",img[idx + 2]);
      // }
      // // printf("%d ",*bitmap);
      newcolor.rgbRed = img[idx + 0];
      newcolor.rgbGreen = img[idx + 1];
      newcolor.rgbBlue = img[idx + 2];
      // newcolor.rgbReserved = img[idx + 3];

      if(!FreeImage_SetPixelColor(bitmap, x, y, &newcolor))
      { fprintf(stderr, "(%d, %d) Fail...\n", x, y); }

      // pixel+=3;
    }
    // next line
    // bits += pitch;
  }
  //printf("%s \n",PathDest);
  if( FreeImage_Save (FIF_JPEG, bitmap , PathDest , 0 ))
  FreeImage_DeInitialise(); //Cleanup !
}
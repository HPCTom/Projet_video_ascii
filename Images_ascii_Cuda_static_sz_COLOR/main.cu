#include <iostream>
#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <sys/time.h>
#include "omp.h"
#include "FreeImage.h"
#include "lib_ascii.h"
#include "host.cu"
#include "device.cu"

#define BPP 24

// Assemblage video python //
static char nbr_img[100] = {0};                                                           // nombre d'images dans la video
// Mesures temps //
static double start,stop,start_kernel,stop_kernel,cpu_time_used,
temps_kernel0_moyen,temps_kernel1_moyen,temps_kernel2_moyen,temps_kernel3_moyen,
temps_kernel4_moyen;                                                                      // variables pour mesurer le temps
static double *temps_kernel0,*temps_kernel1,*temps_kernel2,*temps_kernel3,*temps_kernel4; // pointeur tableau qui stock le temps des kernels à chaque itérations
static char num[10];                                                                      // pour le numéro des iamges (framex.png)
// Barre de chargement //
static char barre[200] = "Traitement ascii des images";                                   //
static float eps;                                                                         // pourcentage équivalent à 1 '#' dans la barre de chargement
static int taille;                                                                        //
static float max_it;                                                                      //
// Pour le traitement d'images //
static unsigned int blockDim_x;                                                           //
static unsigned int blockDim_y;                                                           //
static unsigned int gridDim_x;                                                            //
static unsigned int gridDim_y;                                                            // 
static unsigned int nb_characters;                                                        //
static unsigned int width;                                                                //
static unsigned int height;                                                               //
static unsigned int width_color;                                                          //
static unsigned int height_color;                                                         //
// static unsigned int width_lib;                                                         //
// static unsigned int height_lib;                                                        //
static unsigned int pitch;                                                                //
static unsigned int pitch_final;   
static long unsigned int sz_in_bytes;                                                     // nb de bytes pour l'image initiale
static long unsigned int sz_in_bytes_img_ascii;                                           // nb de bytes pour l'image moyennée
static long unsigned int sz_in_bytes_ascii_color;                                         //

static unsigned int *img;                                                                 // pointeur
static float *img_ascii;                                                                  // pointeur
static unsigned char *img_ascii_color_final;                                              //
static unsigned int *d_img;                                                               // pointeur
static const char *use_ascii;                                                             // pointeur du tableau des ascii qui seront utilisés dans l'image (trié par ordre de niveau de gris croissant)
static unsigned char *d_tab_ascii_lib;                                                    //
float *d_img_ascii;                                                                       // pointeur
static unsigned char *d_img_ascii_color_final;                                            //

static unsigned int gridDim_x_ascii;                                                      // largeur de l'image en nombre d'ascii
static unsigned int gridDim_y_ascii;                                                      //
static unsigned int blockDim_x_ascii;                                                     //
static unsigned int blockDim_y_ascii;                                                     //
static unsigned int nb_sleep_thread_x;                                                    //
static unsigned int nb_sleep_thread_y;                                                    //
static unsigned int nb_sleep_thread_x_ascii;                                              //
static unsigned int nb_sleep_thread_y_ascii;                                              //
static unsigned int nb_sleep_thread_x_color;                                              //
static unsigned int nb_sleep_thread_y_color;                                              //

static unsigned int blockDim_x_color;                                                     //
static unsigned int blockDim_y_color;                                                     //
static unsigned int gridDim_x_color;                                                      //
static unsigned int gridDim_y_color;                                                      //

FIBITMAP *bitmap_final;

float *temp_img;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main (int argc , char** argv)
{

   // ARG_ERROR(argc,argv[1],argv[2]);

   printf("\n[------------------ Réordonnacement du tableau ASCII utilisé pour générer l'image (ordre croissant en niveau de gris) ------------------]\n\n");
   // ############ Librairie ascii ############
   use_ascii = "@8&0a2{+*! ";
   // use_ascii = "8$&03421*! +@{a";
   nb_characters = strlen(use_ascii);
   lib_ascii ascii{use_ascii, use_ascii + nb_characters};
   affiche_ascii(ascii);
   hipMalloc((void**)&d_tab_ascii_lib,ascii.RawSize());
   hipMemcpy(d_tab_ascii_lib,ascii.CaracterArray(),ascii.RawSize(),hipMemcpyHostToDevice);
   gpuErrchk( hipPeekAtLastError() );
   // #########################################

   printf("[------------------ TRAITEMENT ASCII DES IMAGES ------------------]\n\n");
   system("rm -r images_ascii/");
   system("mkdir images_ascii");

   FILE * f_img = popen("find images -type f | wc -l","r");
   fgets(nbr_img, 100, f_img); // calcul le nombre d'image à transformer
   pclose(f_img);

   // Pour la barre de chargement
   eps = 1.5; // pourcentage équivalent à 1 '#' dans la barre
   taille = 0;
   max_it = atoi(nbr_img);
   init_barre_chargement(barre,&taille,eps,max_it);

   temps_kernel0 = (double*) malloc(max_it*sizeof(double));
   temps_kernel1 = (double*) malloc(max_it*sizeof(double));
   temps_kernel2 = (double*) malloc(max_it*sizeof(double));
   temps_kernel3 = (double*) malloc(max_it*sizeof(double));
   temps_kernel4 = (double*) malloc(max_it*sizeof(double));

   start = get_time();
   for(int k=0; k<max_it;k++){

      barre_chargement(barre,100*(k+1)/max_it,k+1,max_it,eps,taille);

      char PathName[100] = "images/frame";
      sprintf(num, "%d", k);
      strcat(PathName, num);
      strcat(PathName,".jpg");

      FreeImage_Initialise();

      // load and decode a regular file
      FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);
      FIBITMAP* bitmap = FreeImage_Load(FIF_JPEG, PathName, 0);


      declaration_1(bitmap,&width,&height,&pitch);
      sz_in_bytes = sizeof(unsigned int) * 3 * width * height; //nb de valeurs pour toute image
      img = (unsigned int*) malloc(sz_in_bytes);
      hipMalloc((void**)&d_img, sz_in_bytes);
      gpuErrchk( hipPeekAtLastError() );

      start_kernel = get_time();
      REORDER_IMG(img,height,width,pitch,bitmap);
      stop_kernel = get_time();
      temps_kernel0[k] = stop_kernel-start_kernel;

      hipMemcpy(d_img, img, sz_in_bytes,hipMemcpyHostToDevice);
      gpuErrchk( hipPeekAtLastError() );

      // //##############################################################################################
      // //################################### Traitement ascii d'image #################################
      // //##############################################################################################


      declaration_2(width,height,&blockDim_x,&blockDim_y,&gridDim_x,&gridDim_y,
                    &blockDim_x_ascii,&blockDim_y_ascii,&gridDim_x_ascii,&gridDim_y_ascii,
                    &nb_sleep_thread_x,&nb_sleep_thread_y,
                    &nb_sleep_thread_x_ascii,&nb_sleep_thread_y_ascii,
                    atof(argv[1]));
      sz_in_bytes_img_ascii = sizeof(float)*gridDim_x_ascii*gridDim_y_ascii;
      img_ascii = (float*) malloc(sz_in_bytes_img_ascii);
      hipMalloc((void**)&d_img_ascii, 4*sz_in_bytes_img_ascii);
      gpuErrchk( hipPeekAtLastError() );
      // printf("\n\n GENERAL : \n blockDim_x = %d \n blockDim_y = %d \n gridDim_x = %d \n gridDim_y = %d \n nb_sleep_thread_x = %d \n nb_sleep_thread_y = %d\n",blockDim_x,blockDim_y,gridDim_x,gridDim_y,nb_sleep_thread_x,nb_sleep_thread_y);
      // printf("\n ASCII : \n blockDim_x_ascii = %d \n blockDim_y_ascii = %d \n gridDim_x_ascii = %d \n gridDim_y_ascii = %d \n nb_sleep_thread_x_ascii = %d \n nb_sleep_thread_y_ascii = %d\n",blockDim_x_ascii,blockDim_y_ascii,gridDim_x_ascii,gridDim_y_ascii,nb_sleep_thread_x_ascii,nb_sleep_thread_y_ascii);
      // printf("Le premier kernel lance une grille de taille %dx%d avec des blocks de taille %dx%d \n",gridDim_x,gridDim_y,blockDim_x,blockDim_y);
      // printf("Le premier kernel doit prendre en compte une sous-grille de taille %dx%d avec des blocks de taille %dx%d \n",gridDim_x_ascii,gridDim_y_ascii,blockDim_x_ascii,blockDim_y_ascii);

      hipMemset(d_img_ascii,0.,4*sz_in_bytes_img_ascii);                                                

      gpuErrchk( hipPeekAtLastError() );

      dim3 dimBlock(blockDim_x,blockDim_y,1);
      dim3 dimGrid(gridDim_x,gridDim_y,1);
      start_kernel = get_time();
      Niveau_Gris_Color_Moyennage<<<dimGrid, dimBlock>>>(d_img_ascii,d_img,width,nb_sleep_thread_x,nb_sleep_thread_y,nb_sleep_thread_x_ascii,
                                                         nb_sleep_thread_y_ascii,gridDim_x_ascii,gridDim_y_ascii,blockDim_x_ascii,blockDim_y_ascii,k);
      stop_kernel = get_time();
      temps_kernel1[k] = stop_kernel-start_kernel;
      gpuErrchk( hipPeekAtLastError() );

      blockDim_x_color = ascii.kwidthCaracter; // 7 
      blockDim_y_color = ascii.kheightCaracter; // 11
      nb_sleep_thread_x_color = nb_sleep_thread_x_ascii;
      nb_sleep_thread_y_color = nb_sleep_thread_y_ascii;
      declaration_3(blockDim_x_color,blockDim_y_color,&gridDim_x_color,&gridDim_y_color,gridDim_x_ascii,gridDim_y_ascii,&sz_in_bytes_ascii_color,
                    &width_color,&height_color);
      // printf("\n\n GENERAL : \n blockDim_x_color = %d \n blockDim_y_color = %d \n gridDim_x_color = %d \n gridDim_y_color = %d \n nb_sleep_thread_x_color = %d \n nb_sleep_thread_y_color = %d\n",blockDim_x_color,blockDim_y_color,gridDim_x_color,gridDim_y_color,nb_sleep_thread_x_color,nb_sleep_thread_y_color);
      
      img_ascii_color_final = (unsigned char*) malloc(sz_in_bytes_ascii_color);
      hipMalloc((void **)&d_img_ascii_color_final, sz_in_bytes_ascii_color); 
      bitmap_final = FreeImage_Allocate(width_color,height_color, BPP);
      pitch_final  = FreeImage_GetPitch(bitmap_final);
 
    
      dim3 dimBlock_color(blockDim_x_color,blockDim_y_color,1);
      dim3 dimGrid_color(gridDim_x_color,gridDim_y_color,1);   
      
      start_kernel = get_time();
      Image_Color<<<dimGrid_color, dimBlock_color>>>(d_img_ascii,d_img_ascii_color_final,d_tab_ascii_lib,width_color,ascii.kwidthCaracter,
                                                     ascii.kheightCaracter,nb_characters,gridDim_x_ascii,gridDim_y_ascii,blockDim_x_ascii,blockDim_y_ascii,
                                                     nb_sleep_thread_x_color,nb_sleep_thread_y_color,atoi(argv[2]),atoi(argv[3]));
      stop_kernel = get_time();
      temps_kernel2[k] = stop_kernel-start_kernel; 
      
      gpuErrchk( hipPeekAtLastError() ); 
      
      start_kernel = get_time();
      hipMemcpy(img_ascii_color_final, d_img_ascii_color_final, sz_in_bytes_ascii_color, hipMemcpyDeviceToHost); 
      stop_kernel = get_time();
      temps_kernel3[k] = stop_kernel-start_kernel; 
      
      gpuErrchk( hipPeekAtLastError() );
      
      start_kernel = get_time();
      SAVE_IMG(img_ascii_color_final,height_color,width_color,pitch_final,bitmap_final,k);  // Créer le pitch&bitmap de img_ascii_color
      stop_kernel = get_time();
      temps_kernel4[k] = stop_kernel-start_kernel;

      free(img);
      free(img_ascii);
      free(img_ascii_color_final);
      hipFree(d_img);
      hipFree(d_img_ascii);
      hipFree(d_img_ascii_color_final);

   }
   stop = get_time();
   cpu_time_used = stop-start;
   hipFreeAsync(d_tab_ascii_lib,0);

   printf("\nTemps pour le traitement d'images : %f secondes\n",cpu_time_used);
   printf("\nTemps moyen pour le traitement d'images : %f secondes\n",cpu_time_used/max_it);
   temps_kernel0_moyen = 0, temps_kernel1_moyen = 0,temps_kernel2_moyen = 0, temps_kernel3_moyen = 0,temps_kernel4_moyen = 0;
   for(int k=0; k<max_it;k++){
      temps_kernel0_moyen += temps_kernel0[k];
      temps_kernel1_moyen += temps_kernel1[k];
      temps_kernel2_moyen += temps_kernel2[k];
      temps_kernel3_moyen += temps_kernel3[k];
      temps_kernel4_moyen += temps_kernel4[k];
   }
   printf("\nTemps moyen pour l'agencement des images brutes : %f secondes\n",temps_kernel0_moyen/max_it);
   printf("Temps moyen pour le traitement du kernel \"Niveau_Gris_Color_Moyennage\": %f secondes\n",temps_kernel1_moyen/max_it);
   printf("Temps moyen pour le traitement du kernel \"Image_Color\" : %f secondes\n",temps_kernel2_moyen/max_it);
   printf("Temps la copie GPU -> CPU : %f secondes\n",temps_kernel3_moyen/max_it);
   printf("Temps moyen pour la création de l'image ascii : %f secondes\n\n",temps_kernel4_moyen/max_it);


   return 0;
}

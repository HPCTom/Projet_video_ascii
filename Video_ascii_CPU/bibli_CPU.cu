
#include <hip/hip_runtime.h>

// Mesures temps
double start,stop,start_kernel,stop_kernel,cpu_time_used,temps_kernel_moyen;
double *temps_kernel;
char num[10]; // Pour le num de framex.png

// Pour la barre de chargement
float p_avant = 0.;
float eps = 1.5; // pourcentage pour 1 '#' dans la barre
char barre[200];


unsigned width;
unsigned height;
unsigned pitch;


int sz_in_bytes;
unsigned int *img;

// Pour le traitement ascii
int block_x;
int grid_x;
int block_y;
int grid_y;

int DETAIL; //nombre d'ascii différents

int gridDim_x; // largeur de l'image en nombre d'ascii
int gridDim_y; // hauteur de l'image en nombre d'ascii

int blockDim_x; // nombre de pixel en largeur qui sera contenu dans 1 ascii
int blockDim_y; // nombre de pixel en hauteur qui sera contenu dans 1 ascii
int blockIdx_x; // Id des blocks en largeur (identique en Cuda)
int blockIdx_y; // Id des blocks en hauteur (identique en Cuda)

int threadIdx_y;
int threadIdx_x;

int n_x;
int n_y;

int nb_sleep_thread_x; // nombres de threads inatif par ligne
int nb_sleep_thread_y; // nombre de threads inactif par colonne

float *img_ascii;

int MAX;

char *final_ascii;

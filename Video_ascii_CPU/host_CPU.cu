#include "hip/hip_runtime.h"
//##############################################################################################
//################################### FONCTION HOST ############################################
//##############################################################################################

//#### Gestion des erreurs dans les parametres d'entré #####
__host__ void error_msg(int argc,int argv1, int argv2, int width){
	if(argc != 4){
		printf("\nUsage : le programme prend 3 arguments.\n"
					 "argv[1] = pourcentage de résolution de l'image entre 0 et 100 (100 %% indique qu'il y aura autant d'ascii en largeur que de pixels).\n"
					 "argv[2] = nombre d'ascii différents utilisés pour générer l'image.\n"
           "argv[3] = nom de la video.\n"
           "Exemple: ./modif_img 50 10 my_video.mp4 \n\n");
	}

	if(argv1 > 100 || argv1 <= 0){
		printf("\nLe pourcentage de résolution de l'image doit etre compris entre 0 (exclu) et 100 (cf README).\n\n");
	}
}

//#### Declaration des parametres
__host__ void declaration_1(FIBITMAP *bitmap,unsigned *width,unsigned *height,unsigned *pitch,unsigned int *img,int argc,int argv1, int argv2){
    *width  = FreeImage_GetWidth(bitmap);
    *height = FreeImage_GetHeight(bitmap);
    *pitch  = FreeImage_GetPitch(bitmap);
    error_msg(argc,argv1,argv2,*width); // gestion des erreur les arguments d'entrée
}

__host__ void declaration_2(int *DETAIL, int *gridDim_x,int *gridDim_y,int *blockDim_x,int *blockDim_y,int width, int height,
                            int *n_x,int *n_y,int *nb_sleep_thread_x,int *nb_sleep_thread_y,float argv3, int argv4){
  *DETAIL = argv4; //nombre d'ascii différents

  *gridDim_x = (int)((float)width*argv3/100.); // largeur de l'image en nombre d'ascii

  float ratio = (float)width/(float)height*1.8; //calcul la hauteur de l'image en prennant en compte le ratio de l'image Hauteur/Largeur et compense le fait que les characteres soit plus espacés en hauteur que en largeur (1.8) dans un fihcier texte.
	*gridDim_y = *gridDim_x/ratio; // hauteur de l'image en nombre d'ascii

  *blockDim_x = ceil((float)width/(float)(*gridDim_x));
  *blockDim_y = ceil((float)height/(float)(*gridDim_y));

  int nb_sleep_thread_x_all = *gridDim_x*(*blockDim_x)-width; // nombres de threads inactif selon x
  int nb_sleep_thread_y_all = *gridDim_y*(*blockDim_y)-height; // nombres de threads inactif selon y
  int nb_sleep_block_x = nb_sleep_thread_x_all/(*blockDim_x); // nombre de blocks inactifs selon x (arrondie inférieur)
  int nb_sleep_block_y = nb_sleep_thread_y_all/(*blockDim_y); // nombre de blocs inactis selon y (arrondie inférieur)

  *gridDim_x = *gridDim_x - nb_sleep_block_x; // nouvelle grid en x redimensionnée
  *gridDim_y = *gridDim_y - nb_sleep_block_y; // nouvelle grid en y redimensionnée

  *n_x = *gridDim_x*(*blockDim_x); // nombre de threads par ligne
  *n_y = *gridDim_y*(*blockDim_y); // nombres de threads par colonne
  *nb_sleep_thread_x = *n_x-width; // nombres de threads inatif par ligne
  *nb_sleep_thread_y = *n_y-height; // nombre de threads inactif par colonne

}


void init_barre_chargement(char *barre,int *cpt,float eps,int max_it){
  strcat(barre, " [");
  while(barre[*cpt]!='\0'){
    *cpt = *cpt+1;
  }
  int taille = (int)ceil(100./eps); //nombre de #

  for(int t=0;t<taille;t++){
    strcat(barre, " ");
  }
  strcat(barre, "]");
}

void barre_chargement(char *barre,float p,int k, float max, float eps,int taille){ //entier entre 0 et 100

  int idx = (int)(p/eps); // a quelle intervalle j'appartiens

  if(k==max){
    for(int k=taille; k<taille+idx+1; k++){
      barre[k] = '#';
    }
    printf("\r%s  %.2f%% \n",barre,p);
    fflush(stdout);
  }
  else{
    for(int k=taille; k<taille+idx; k++){
      barre[k] = '#';
    }
    printf("\r%s  %.2f%% ",barre,p);
    fflush(stdout);
  }
}

//#### Permet de calculer le temps #####
__host__ double get_time() {
  struct timeval tv;
  gettimeofday(&tv, (void *)0);
  return (double) tv.tv_sec + tv.tv_usec*1e-6;
}

//#### Ordonne le tableau en RGB #####
__host__ void REORDER_IMG(unsigned int *img,int  height, int width,unsigned pitch,FIBITMAP* bitmap){
	BYTE *bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = bits;
    for ( int x =0; x<width; x++)
    {
      int idx = ((y * width) + x) * 3;
      img[idx + 0] = pixel[FI_RGBA_RED];
      img[idx + 1] = pixel[FI_RGBA_GREEN];
      img[idx + 2] = pixel[FI_RGBA_BLUE];
      pixel += 3;
    }
    bits += pitch;
  }
}

//#### Sauvergarde l'image finale #####
__host__ void SAVE_IMG(unsigned int *img,int  height, int width,const char *PathDest,unsigned pitch,FIBITMAP* bitmap){
	BYTE* bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = (BYTE*)bits;
    for ( int x =0; x<width; x++)
    {
      RGBQUAD newcolor;

      int idx = ((y * width) + x) * 3;
      newcolor.rgbRed = img[idx + 0];
      newcolor.rgbGreen = img[idx + 1];
      newcolor.rgbBlue = img[idx + 2];

      if(!FreeImage_SetPixelColor(bitmap, x, y, &newcolor))
      { fprintf(stderr, "(%d, %d) Fail...\n", x, y); }

      pixel+=3;
    }
    // next line
    bits += pitch;
  }

  if( FreeImage_Save (FIF_PNG, bitmap , PathDest , 0 ))
  FreeImage_DeInitialise(); //Cleanup !
}

__host__ void choix_ascii(float *img_ascii,int taille,int taille_x,int taille_y,char *tab_final,
													int min, int DETAIL, int nb_sleep_thread_x, int nb_sleep_thread_y){

	char ascii[255] = {'8','*','0','w','^','&','=','!','$','4','q','+','1','m','#','%','l',':','2','<','>','}','5','/','.','2','a','3','p','t','6','?','9','c','7','r','[',']','x','b'}; //40
  //char ascii[255] = {'8','&','4','w','^','*','=','!','$','0','q','+','1','m','#','%','l',':','2','<','>','}','5','/','.','2','a','3','p','t','6','?','9','c','7','r','[',']','x','b'}; //40

	if(DETAIL > 255){
		printf("nombres d'ascii max dépacé DETAIL = %d et MAX = %d\n",DETAIL,255);
	}

	int eps = 255/DETAIL;
	int moy;

	for(int i=0;i<taille;i++)
	{
		moy = img_ascii[i];
		if(moy/eps > DETAIL-1){
			tab_final[i] = ascii[moy/eps-1];
		}
		else{
			tab_final[i] = ascii[moy/eps];
		}
	}

}

__host__ void tab_to_txt(char *final_ascii,float *img_ascii,char *tab,int hauteur_ascii,int largeur_ascii,
	 											 int taille_x, int taille_y, int DETAIL, int nb_sleep_thread_x, int nb_sleep_thread_y)
{
	FILE *fp = NULL;
	fp = fopen(tab,"w");

	if(fp ==NULL)
	{
		printf("\ntab_to_txt : ERREUR OUVERTURE FICHIER\n");
	}

	choix_ascii(img_ascii,largeur_ascii*hauteur_ascii,taille_x,taille_y,final_ascii,0,DETAIL,nb_sleep_thread_x,nb_sleep_thread_y);

	int cpt = 0;

	for(int k=largeur_ascii*hauteur_ascii-1;k>-1;k--){

		if(cpt == largeur_ascii){
			cpt = 0;
			fprintf(fp,"\n");
		}
		cpt = cpt+1;
		if(k>0){
			fprintf(fp,"%c",final_ascii[k]);
		}
		else{
			fprintf(fp,"%c",final_ascii[0]);
		}
	}
	fclose(fp);

}

__host__ void txt_to_png(int width,char* tab_txt, char* tab_png)
{

	char ligne[width];

	sprintf(ligne,"convert -font Courier -background white -fill black label:@%s -flatten images_ascii/%s",tab_txt,tab_png);

  system(ligne);
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <sys/time.h>
#include "FreeImage.h"
#include "bibli_CPU.cu"
#include "host_CPU.cu"

int main (int argc , char** argv)
{

  printf("\n[------------------ DECOUPAGE DE LA VIDEO ------------------]\n");
  char decoupe[100] = "python3 decoupe_vid.py ";
  strcat(decoupe, argv[3]);
  system(decoupe);

  printf("[------------------ TRAITEMENT ASCII DES IMAGES ------------------]\n\n");
  system("rm -r images_ascii/");
  system("mkdir images_ascii");

  FILE * f_img = popen("find images -type f | wc -l","r");
  char nbr_img[100] = {0};
  fgets(nbr_img, 100, f_img); // calcul le nombre d'image à transformer
  pclose(f_img);

  // Pour la barre de chargement
  float eps = 1.5; // pourcentage équivalent à 1 '#' dans la barre
  int taille = 0;
  char barre[200] = "Traitement ascii des images";
  float max_it = atoi(nbr_img);
  init_barre_chargement(barre,&taille,eps,max_it);

  temps_kernel = (double*) malloc(max_it*sizeof(double));

  start = get_time();
  for(int k=0; k<max_it;k++){

    barre_chargement(barre,100*(k+1)/max_it,k+1,max_it,eps,taille);

    char PathName[100] = "images/frame";
    sprintf(num, "%d", k);
    strcat(PathName, num);
    strcat(PathName,".jpg");

    FreeImage_Initialise();

    // load and decode a regular file
    FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);
    FIBITMAP* bitmap = FreeImage_Load(FIF_JPEG, PathName, 0);

    if(k==0){
      declaration_1(bitmap,&width,&height,&pitch,img,argc,atoi(argv[1]),atoi(argv[2]));
      sz_in_bytes = sizeof(unsigned int) * 3 * width * height; //nb de valeurs pour toute image
      img = (unsigned int*) malloc(sz_in_bytes);
    }

    REORDER_IMG(img,height,width,pitch,bitmap);

    //##############################################################################################
    //################################### Traitement ascii d'image #################################
    //##############################################################################################

    if(k==0){
      declaration_2(&DETAIL, &gridDim_x,&gridDim_y,&blockDim_x,&blockDim_y,width,height,
      &n_x,&n_y,&nb_sleep_thread_x,&nb_sleep_thread_y,atof(argv[1]),atoi(argv[2]));
    }

    img_ascii = (float*) calloc(sizeof(float),gridDim_x*gridDim_y);

    threadIdx_y = 0;
    threadIdx_x = 0;

    start_kernel = get_time();
    for(int h=0;h<gridDim_y*blockDim_y;h++){
      blockIdx_y = floor(h/blockDim_y);
      for(int w=0;w<gridDim_x*blockDim_x;w++){
        blockIdx_x = floor(w/blockDim_x);

        if(blockIdx_y < gridDim_y-1 || blockIdx_y == gridDim_y-1 && threadIdx_y < blockDim_y-nb_sleep_thread_y){
		      if(blockIdx_x < gridDim_x-1 || blockIdx_x == gridDim_x-1 && threadIdx_x < blockDim_x-nb_sleep_thread_x){

            int x = blockIdx_x * blockDim_x + threadIdx_x;
            int y = blockIdx_y * blockDim_y + threadIdx_y;

            int idx = ((y * width) + width - x) * 3;
            int idx_ascii  = gridDim_x*blockIdx_y + blockIdx_x;

            if(blockIdx_y == gridDim_y-1){
              img_ascii[idx_ascii] += (0.299*img[idx+0]+0.587*img[idx+1]+0.114*img[idx+2])/(blockDim_x*(blockDim_y-nb_sleep_thread_y));
			      }
            else if(blockIdx_x == gridDim_x-1){
              img_ascii[idx_ascii] += (0.299*img[idx+0]+0.587*img[idx+1]+0.114*img[idx+2])/((blockDim_x-nb_sleep_thread_x)*blockDim_y);
            }
            else{
              img_ascii[idx_ascii] += (0.299*img[idx+0]+0.587*img[idx+1]+0.114*img[idx+2])/(blockDim_x*blockDim_y);
            }

		      }
	      }
        threadIdx_x += 1;
        if(blockDim_x == threadIdx_x){
          threadIdx_x = 0;
        }
      }
      threadIdx_y += 1;
      if(blockDim_y == threadIdx_y){
          threadIdx_y = 0;
      }
    }
    stop_kernel = get_time();
    temps_kernel[k] = stop_kernel-start_kernel;

    char tab_txt[100]; // nom de l'image txt de sortie
    char tab_png[100]; // nom de l'image png de sortie
    strcat(strcpy(tab_txt, "autre/temporaire/frame"), num);
    strcat(strcpy(tab_png, "frame"), num);
    strcat(tab_txt, ".txt");
    strcat(tab_png, ".png");

    if(k==0){
      final_ascii = (char*) malloc(sizeof(char)*gridDim_x*gridDim_y);
    }

    tab_to_txt(final_ascii,img_ascii,tab_txt,gridDim_y,gridDim_x,blockDim_x,blockDim_y,DETAIL,nb_sleep_thread_x,nb_sleep_thread_y); // Creation du fichier texte contenant l'image

    txt_to_png(width,tab_txt,tab_png); // Creation de l'image png à l'aide du fichier txt créé precedement
    free(img_ascii);
  }
  stop = get_time();
  cpu_time_used = stop-start;
  free(img);
  free(final_ascii);

  printf("\nTemps pour le traitement d'images : %f secondes\n",cpu_time_used);
  temps_kernel_moyen = 0;
  for(int k=0; k<max_it;k++){
    temps_kernel_moyen += temps_kernel[k];
  }
  printf("\nTemps moyen pour le traitement d'un kernel : %f secondes\n\n",temps_kernel_moyen/max_it);

  printf("[------------------ ASSEMBLAGE DE LA VIDEO ------------------]\n\n");
  char assemble[100] = "python3 assemble_vid.py ";
  strcat(assemble, argv[3]);
  system(assemble);

  printf("\nVideo bien assemblée.\n");

  return 0;
}
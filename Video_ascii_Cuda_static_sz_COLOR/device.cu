
#include <hip/hip_runtime.h>
//##############################################################################################
//################################### FONCTION DEVICE ##########################################
//##############################################################################################


__global__ void Init_float_num(float *d_img, unsigned int width, unsigned int height,unsigned int nb_sleep_thread_x,
							   unsigned int nb_sleep_thread_y,float num)
{

	// dim3 dimBlock_init(blockDim_x_ascii,blockDim_y_ascii,1);
    // dim3 dimGrid_init(gridDim_x_ascii,gridDim_y_ascii,1);
    // Init_float_num<<<dimGrid_init, dimBlock_init>>>(d_img_ascii,gridDim_x_ascii,gridDim_y_ascii,
    //                                                 nb_sleep_thread_x_ascii,nb_sleep_thread_y_ascii,0.);

	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){
			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			int idx = ((y * width) + x) * 4;

			d_img[idx+0] = num;
			d_img[idx+1] = num;	
			d_img[idx+2] = num;	
			d_img[idx+3] = num;
			
		}
	}
}



__global__ void Niveau_Gris_Color_Moyennage(float *d_img_ascii,unsigned int* d_img,unsigned width,
											int nb_sleep_thread_x,int nb_sleep_thread_y,
 									  		int nb_sleep_thread_x_ascii,int nb_sleep_thread_y_ascii,
									  		int gridDim_x_ascii,int gridDim_y_ascii,int blockDim_x_ascii,
											int blockDim_y_ascii,int ite)
{
	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){

			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			int idx = ((y * width) + width - x) * 3;

			int idx_ascii = (int)((float)y/(float)blockDim_y_ascii) * gridDim_x_ascii + (int)((float)x/(float)blockDim_x_ascii);

			atomicAdd(&d_img_ascii[4*idx_ascii+0],0.299*d_img[idx+0]+0.587*d_img[idx+1]+0.114*d_img[idx+2]);
			atomicAdd(&d_img_ascii[4*idx_ascii+1],d_img[idx+0]);
			atomicAdd(&d_img_ascii[4*idx_ascii+2],d_img[idx+1]);
			atomicAdd(&d_img_ascii[4*idx_ascii+3],d_img[idx+2]);

		}
	}

}


// kernel avec des blocks en ascii.kwidthCaracter*(ascii.kheightCaracter+1)
__global__ void Image_Color(float *d_img_ascii,unsigned char *d_img_ascii_color_final,unsigned char *d_tab_ascii_lib,
							unsigned int width_color,unsigned int width_lib,unsigned int height_lib,
							unsigned int nb_characters,unsigned int gridDim_x_ascii,unsigned int gridDim_y_ascii,
							int blockDim_x_ascii,int blockDim_y_ascii,unsigned int nb_sleep_thread_x_color, unsigned int nb_sleep_thread_y_color,
							int mode_a, int mode_b)
{

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = ((y * width_color) + width_color - x) * 3 - 3;
	int idx_ascii = blockIdx.y * gridDim.x + blockIdx.x;
	float eps = 0.1;

	if(blockIdx.x == gridDim.x-1 && blockIdx.y == gridDim.y-1){
		int deno = (blockDim_x_ascii-nb_sleep_thread_x_color)*(blockDim_y_ascii-nb_sleep_thread_y_color);
		int pos = (int)(nb_characters*(((float)d_img_ascii[4*idx_ascii+0]/(deno))/(float)255));
		int idx_lib = pos*width_lib*height_lib;
		int idx_tab_ascii_lib = idx_lib + width_lib*height_lib - (threadIdx.y+1)*blockDim.x + width_lib-threadIdx.x-1;
		float val_red = (d_img_ascii[4*idx_ascii+1])/(deno);
		float val_green = (d_img_ascii[4*idx_ascii+2])/(deno);
		float val_blue = (d_img_ascii[4*idx_ascii+3])/(deno);
		if(d_tab_ascii_lib[idx_tab_ascii_lib] == 0){
			int var_temp_b = __float2int_rn(mode_b/255); // 0 si 0 ou 1 sinon 1 pour 255
			float background_red = (val_red+eps)*((((mode_b*var_temp_b)/(val_red+eps))+abs((var_temp_b-1)*mode_b)));
			float background_green = (val_green+eps)*((((mode_b*var_temp_b)/(val_green+eps))+abs((var_temp_b-1)*mode_b)));
			float background_blue = (val_blue+eps)*((((mode_b*var_temp_b)/(val_blue+eps))+abs((var_temp_b-1)*mode_b)));

			d_img_ascii_color_final[idx+0] = __float2int_rn(background_red);
			d_img_ascii_color_final[idx+1] = __float2int_rn(background_green);
			d_img_ascii_color_final[idx+2] = __float2int_rn(background_blue);
		}
		else{
			int var_temp_a = __float2int_rd(mode_a/255); // 0 si 0 ou 1 sinon 1 pour 255
			float ascii_red = (val_red+eps)*((((mode_a*var_temp_a)/(val_red+eps))+abs((var_temp_a-1)*mode_a)));
			float ascii_green = (val_green+eps)*((((mode_a*var_temp_a)/(val_green+eps))+abs((var_temp_a-1)*mode_a)));
			float ascii_blue = (val_blue+eps)*((((mode_a*var_temp_a)/(val_blue+eps))+abs((var_temp_a-1)*mode_a)));

			d_img_ascii_color_final[idx+0] = __float2int_rn(ascii_red);
			d_img_ascii_color_final[idx+1] = __float2int_rn(ascii_green);
			d_img_ascii_color_final[idx+2] = __float2int_rn(ascii_blue);
		}
	}

	else if(blockIdx.x == gridDim.x-1){
		int deno = (blockDim_x_ascii-nb_sleep_thread_x_color)*blockDim_y_ascii;
		int pos = (int)(nb_characters*(((float)d_img_ascii[4*idx_ascii+0]/(deno))/(float)255));
		int idx_lib = pos*width_lib*height_lib;
		int idx_tab_ascii_lib = idx_lib + width_lib*height_lib - (threadIdx.y+1)*blockDim.x + width_lib-threadIdx.x-1;
		float val_red = (d_img_ascii[4*idx_ascii+1])/(deno);
		float val_green = (d_img_ascii[4*idx_ascii+2])/(deno);
		float val_blue = (d_img_ascii[4*idx_ascii+3])/(deno);
		if(d_tab_ascii_lib[idx_tab_ascii_lib] == 0){
			int var_temp_b = __float2int_rn(mode_b/255); // 0 si 0 ou 1 sinon 1 pour 255
			float background_red = (val_red+eps)*((((mode_b*var_temp_b)/(val_red+eps))+abs((var_temp_b-1)*mode_b)));
			float background_green = (val_green+eps)*((((mode_b*var_temp_b)/(val_green+eps))+abs((var_temp_b-1)*mode_b)));
			float background_blue = (val_blue+eps)*((((mode_b*var_temp_b)/(val_blue+eps))+abs((var_temp_b-1)*mode_b)));

			d_img_ascii_color_final[idx+0] = __float2int_rn(background_red);
			d_img_ascii_color_final[idx+1] = __float2int_rn(background_green);
			d_img_ascii_color_final[idx+2] = __float2int_rn(background_blue);
		}
		else{
			int var_temp_a = __float2int_rd(mode_a/255); // 0 si 0 ou 1 sinon 1 pour 255
			float ascii_red = (val_red+eps)*((((mode_a*var_temp_a)/(val_red+eps))+abs((var_temp_a-1)*mode_a)));
			float ascii_green = (val_green+eps)*((((mode_a*var_temp_a)/(val_green+eps))+abs((var_temp_a-1)*mode_a)));
			float ascii_blue = (val_blue+eps)*((((mode_a*var_temp_a)/(val_blue+eps))+abs((var_temp_a-1)*mode_a)));

			d_img_ascii_color_final[idx+0] = __float2int_rn(ascii_red);
			d_img_ascii_color_final[idx+1] = __float2int_rn(ascii_green);
			d_img_ascii_color_final[idx+2] = __float2int_rn(ascii_blue);
		}
	}

	else if(blockIdx.y == gridDim.y-1){
		int deno = blockDim_x_ascii*(blockDim_y_ascii-nb_sleep_thread_y_color);
		int pos = (int)(nb_characters*(((float)d_img_ascii[4*idx_ascii+0]/(deno))/(float)255));
		int idx_lib = pos*width_lib*height_lib;
		int idx_tab_ascii_lib = idx_lib + width_lib*height_lib - (threadIdx.y+1)*blockDim.x + width_lib-threadIdx.x-1;
		float val_red = (d_img_ascii[4*idx_ascii+1])/(deno);
		float val_green = (d_img_ascii[4*idx_ascii+2])/(deno);
		float val_blue = (d_img_ascii[4*idx_ascii+3])/(deno);
		if(d_tab_ascii_lib[idx_tab_ascii_lib] == 0){
			int var_temp_b = __float2int_rn(mode_b/255); // 0 si 0 ou 1 sinon 1 pour 255
			float background_red = (val_red+eps)*((((mode_b*var_temp_b)/(val_red+eps))+abs((var_temp_b-1)*mode_b)));
			float background_green = (val_green+eps)*((((mode_b*var_temp_b)/(val_green+eps))+abs((var_temp_b-1)*mode_b)));
			float background_blue = (val_blue+eps)*((((mode_b*var_temp_b)/(val_blue+eps))+abs((var_temp_b-1)*mode_b)));

			d_img_ascii_color_final[idx+0] = __float2int_rn(background_red);
			d_img_ascii_color_final[idx+1] = __float2int_rn(background_green);
			d_img_ascii_color_final[idx+2] = __float2int_rn(background_blue);
		}
		else{
			int var_temp_a = __float2int_rd(mode_a/255); // 0 si 0 ou 1 sinon 1 pour 255
			float ascii_red = (val_red+eps)*((((mode_a*var_temp_a)/(val_red+eps))+abs((var_temp_a-1)*mode_a)));
			float ascii_green = (val_green+eps)*((((mode_a*var_temp_a)/(val_green+eps))+abs((var_temp_a-1)*mode_a)));
			float ascii_blue = (val_blue+eps)*((((mode_a*var_temp_a)/(val_blue+eps))+abs((var_temp_a-1)*mode_a)));

			d_img_ascii_color_final[idx+0] = __float2int_rn(ascii_red);
			d_img_ascii_color_final[idx+1] = __float2int_rn(ascii_green);
			d_img_ascii_color_final[idx+2] = __float2int_rn(ascii_blue);
		}
	}

	else{
		int deno = blockDim_x_ascii*blockDim_y_ascii; 																		// Nombre de threads ayant participé au caractère
		int pos = (int)(nb_characters*(((float)d_img_ascii[4*idx_ascii+0]/(deno))/(float)255));								// choix du caracter en fonction du niveau de gris
		int idx_lib = pos*width_lib*height_lib; 																			// Position du carcatere dans la lib
		int idx_tab_ascii_lib = idx_lib + width_lib*height_lib - (threadIdx.y+1)*blockDim.x + width_lib-threadIdx.x-1;		// Indice dans la lib de caractères
		float val_red = (d_img_ascii[4*idx_ascii+1])/(deno);																// Couleur rouge moyennée
		float val_green = (d_img_ascii[4*idx_ascii+2])/(deno);																// Couleur verte moyennée
		float val_blue = (d_img_ascii[4*idx_ascii+3])/(deno);																// Couleur bleue moyennée
		if(d_tab_ascii_lib[idx_tab_ascii_lib] == 0){	
			int var_temp_b = __float2int_rn(mode_b/255); 																	// 0 si 0 ou 1, sinon, 1 pour 255
			float background_red = (val_red+eps)*((((mode_b*var_temp_b)/(val_red+eps))+abs((var_temp_b-1)*mode_b)));		// Couleur du fond rouge , complexe mais perlet d'avoir 3 modes différents (noir,blanc et couleur) avec 3 input (0,255 et 1)
			float background_green = (val_green+eps)*((((mode_b*var_temp_b)/(val_green+eps))+abs((var_temp_b-1)*mode_b))); 	// Couleur du fond vert , complexe mais perlet d'avoir 3 modes différents (noir,blanc et couleur) avec 3 input (0,255 et 1)
			float background_blue = (val_blue+eps)*((((mode_b*var_temp_b)/(val_blue+eps))+abs((var_temp_b-1)*mode_b)));		// Couleur du fond bleu, complexe mais perlet d'avoir 3 modes différents (noir,blanc et couleur) avec 3 input (0,255 et 1)

			d_img_ascii_color_final[idx+0] = __float2int_rn(background_red);
			d_img_ascii_color_final[idx+1] = __float2int_rn(background_green);
			d_img_ascii_color_final[idx+2] = __float2int_rn(background_blue);
		}
		else{
			int var_temp_a = __float2int_rd(mode_a/255);
			float ascii_red = (val_red+eps)*((((mode_a*var_temp_a)/(val_red+eps))+abs((var_temp_a-1)*mode_a)));
			float ascii_green = (val_green+eps)*((((mode_a*var_temp_a)/(val_green+eps))+abs((var_temp_a-1)*mode_a)));
			float ascii_blue = (val_blue+eps)*((((mode_a*var_temp_a)/(val_blue+eps))+abs((var_temp_a-1)*mode_a)));

			d_img_ascii_color_final[idx+0] = __float2int_rn(ascii_red);
			d_img_ascii_color_final[idx+1] = __float2int_rn(ascii_green);
			d_img_ascii_color_final[idx+2] = __float2int_rn(ascii_blue);
		}
	}


}
#include "hip/hip_runtime.h"
//##############################################################################################
//################################### FONCTION HOST ############################################
//##############################################################################################

__host__ void error_msg(int msg_case,const char msg){
  switch(msg_case){
    case 0:
      printf("\nUsage : le programme prend au moins 3 arguments.\n"
      "argv[1] = pourcentage de résolution de l'image entre 0 et 100 (100 %% indique qu'il y aura autant d'ascii en largeur que de pixels).\n"
      "argv[2] = nombre d'ascii différents utilisés pour générer l'image.\n"
      "argv[3] = nom de la video.\n"
      "Exemple: ./modif_img 50 10 my_video.mp4 \n\n");
      break;

    case 1:
      printf("ERROR argument invalide : %c \n",msg);
      break;
  }
}

//#### Gestion des erreurs dans les parametres d'entré #####
__host__ void ARG_ERROR(int argc,char argv1,char argv2){
  int nb_max_arg = 5;

  const char *default_msg[4] = {"0", "orange", "yellow", "blue"};

	if(argc > nb_max_arg || argc < 4){
		error_msg(0,*default_msg[0]);
    exit (EXIT_FAILURE);
	}

  else if(atoi(&argv1) <= 0 || atoi(&argv1) > 100){
    error_msg(0,*default_msg[0]);
    error_msg(1,argv1);
    exit (EXIT_FAILURE);
  }

  else if(atoi(&argv2) <= 0 || atoi(&argv2) > 10){
    error_msg(0,*default_msg[0]);
    error_msg(1,argv2);
    exit (EXIT_FAILURE);
  }

  else{
    printf("Pas de problèmes dans les arguments \n");
  }

}

//#### Declaration des parametres
__host__ void declaration_1(FIBITMAP *bitmap,unsigned *width,unsigned *height,unsigned *pitch){
    *width  = FreeImage_GetWidth(bitmap);
    *height = FreeImage_GetHeight(bitmap);
    *pitch  = FreeImage_GetPitch(bitmap);
    // printf("pitch = %d \n",*pitch);
}

__host__ void declaration_2(unsigned int width, unsigned int height,unsigned int *blockDim_x,unsigned int *blockDim_y,unsigned int *gridDim_x,unsigned int *gridDim_y,
                            unsigned int *blockDim_x_ascii,unsigned int *blockDim_y_ascii,unsigned int *gridDim_x_ascii,unsigned int *gridDim_y_ascii,
                            unsigned int *nb_sleep_thread_x,unsigned int *nb_sleep_thread_y,unsigned int *nb_sleep_thread_x_ascii,unsigned int *nb_sleep_thread_y_ascii,
                            float poucrentage_image){                        

  // #### GRILLE GENERALE ####
  *blockDim_x = 16; // ok 
  *blockDim_y = 16; // ok
  *gridDim_x = ceil((float)width/(float)(*blockDim_x)); // ok 
  *gridDim_y = ceil((float)height/(float)(*blockDim_y)); // ok
  *nb_sleep_thread_x = *gridDim_x*(*blockDim_x)-width; // nombres de threads inatif par ligne
  *nb_sleep_thread_y = *gridDim_y*(*blockDim_y)-height; // nombre de threads inactif par colonne

  // #### DIMENSION ASCII FINALE ####
  int nb_ascii_largeur = ceil(((float)width*poucrentage_image/100.));       //nombre d'ascii qu'il y aura en largeur sur l'image finale
  float ratio = (float)width/(float)height*1.5;                             //ratio largeur/hauteur pour calculer la hauteur en ascii de l'image finale (1.5 prend en compte l'écart plus grand entre les lignes que les colonnes dans un fichier texte)
  int nb_ascii_hauteur = ceil(((float)nb_ascii_largeur/ratio));             //nombre d'ascii qu'il y aura en largeur sur l'image finale

  // //### GRILLE SECONDAIRE ###
  int gridDim_x_ascii_all = nb_ascii_largeur; 
	int gridDim_y_ascii_all = nb_ascii_hauteur;

  *blockDim_x_ascii = ceil((float)width/(float)(gridDim_x_ascii_all));
  *blockDim_y_ascii = ceil((float)height/(float)(gridDim_y_ascii_all));

  int nb_sleep_thread_x_ascii_all = gridDim_x_ascii_all*(*blockDim_x_ascii)-width;   // nombres de threads inactif selon x
  int nb_sleep_thread_y_ascii_all = gridDim_y_ascii_all*(*blockDim_y_ascii)-height;  // nombres de threads inactif selon y
  int nb_sleep_block_x_all = (int)((float)nb_sleep_thread_x_ascii_all/(float)(*blockDim_x_ascii)); // nombre de blocks inactifs selon x pour le premier block qui dépasse (arrondie inférieur)
  int nb_sleep_block_y_all = (int)((float)nb_sleep_thread_y_ascii_all/(float)(*blockDim_y_ascii)); // nombre de blocs inactis selon y pour le premier block qui dépasse (arrondie inférieur)

  *gridDim_x_ascii = gridDim_x_ascii_all - nb_sleep_block_x_all; // nouvelle grid en x redimensionnée 
  *gridDim_y_ascii = gridDim_y_ascii_all - nb_sleep_block_y_all; // nouvelle grid en y redimensionnée 

  *nb_sleep_thread_x_ascii = *gridDim_x_ascii*(*blockDim_x_ascii)-width; // nombres de threads inatif par ligne
  *nb_sleep_thread_y_ascii = *gridDim_y_ascii*(*blockDim_y_ascii)-height; // nombre de threads inactif par colonne

}

__host__ void declaration_3(unsigned int blockDim_x_color,unsigned int blockDim_y_color,unsigned int *gridDim_x_color,unsigned int *gridDim_y_color,
                            unsigned int gridDim_x_ascii,unsigned int gridDim_y_ascii, long unsigned int *sz_in_bytes_ascii_color,
                            unsigned int *width_color,unsigned int *height_color){   


  *width_color = blockDim_x_color*gridDim_x_ascii;
  *height_color = blockDim_y_color*gridDim_y_ascii;                          

  *gridDim_x_color = gridDim_x_ascii;
  *gridDim_y_color = gridDim_y_ascii; 

  *sz_in_bytes_ascii_color = 3*blockDim_x_color*blockDim_y_color*gridDim_x_ascii*gridDim_y_ascii*sizeof(unsigned int);

}


void init_barre_chargement(char *barre,int *taille,float *eps,int *max_it,const char *nbr_img){
  *max_it = atoi(nbr_img);
  *taille = 0;
  *eps = 1.5;
  strcat(barre, " [");
  while(barre[*taille]!='\0'){
    *taille = *taille+1;
  }
  int taille_intervalle = (int)ceil(100./(*eps)); //nombre de #

  for(int t=0;t<taille_intervalle;t++){
    strcat(barre, " ");
  }
  strcat(barre, "]");
}

void barre_chargement(char *barre,float p,int k, int max_it, float eps,int taille){ //entier entre 0 et 100

  int idx = (int)(p/eps); // a quelle intervalle j'appartiens

  if(k==max_it){
    for(int k=taille; k<taille+idx+1; k++){
      barre[k] = '#';
    }
    printf("\r%s  %.2f%% \n",barre,p);
    fflush(stdout);
  }
  else{
    for(int k=taille; k<taille+idx; k++){
      barre[k] = '#';
    }
    printf("\r%s  %.2f%% ",barre,p);
    fflush(stdout);
  }
}

__host__ void START_IT(char *barre,int k, int max_it, float eps,int taille,char *PathName, char *num){
  
  barre_chargement(barre,100*(k+1)/max_it,k+1,max_it,eps,taille);
  strcpy(PathName, "images/frame");
  sprintf(num, "%d", k);
  strcat(PathName, num);
  strcat(PathName,".jpg");
}

//#### Permet de calculer le temps #####
__host__ double get_time() {
  struct timeval tv;
  gettimeofday(&tv, (void *)0);
  return (double) tv.tv_sec + tv.tv_usec*1e-6;
}

//#### Ordonne le tableau en RGB? #####
__host__ void REORDER_IMG(unsigned int *img,int  height, int width,unsigned pitch,FIBITMAP* bitmap){
	BYTE *bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = bits;
    for ( int x =0; x<width; x++)
    {
      int idx = ((y * width) + x) * 3;
      img[idx + 0] = pixel[FI_RGBA_RED];
      img[idx + 1] = pixel[FI_RGBA_GREEN];
      img[idx + 2] = pixel[FI_RGBA_BLUE];
      pixel += 3;
    }
    bits += pitch;
  }
  FreeImage_DeInitialise();
}

//#### Sauvergarde l'image finale #####
__host__ void SAVE_IMG(unsigned char *img,unsigned int height,unsigned int width,unsigned int pitch,FIBITMAP* bitmap,int num){

  FreeImage_Initialise();
  int cpt = 1;
  float val = (float)num;
  while (val/10 > 1){
    cpt = cpt+1;
    val = val/10;
  }
  int size_PathDest = 22+cpt;
  char PathDest[size_PathDest];                            // nom de l'image png de sortie
  sprintf(PathDest,"images_ascii/frame%d.png", num);

  RGBQUAD newcolor;

  for ( int y =0; y<height; y++)
  {
    for ( int x =0; x<width; x++)
    {

      int idx = ((y * width) + x) * 3;

      newcolor.rgbRed = img[idx + 0];
      newcolor.rgbGreen = img[idx + 1];
      newcolor.rgbBlue = img[idx + 2];

      if(!FreeImage_SetPixelColor(bitmap, x, y, &newcolor))
      { fprintf(stderr, "(%d, %d) Fail allocation 2...\n", x, y); }

    }
  }
  FreeImage_Save (FIF_PNG, bitmap , PathDest , 0 );
}


__host__ void data_preparation( FIBITMAP **bitmap,unsigned int *width,unsigned int *height,unsigned int *pitch,unsigned long *sz_in_bytes,unsigned int **img,unsigned int **d_img,
                                unsigned int *blockDim_x,unsigned int *blockDim_y,unsigned int* gridDim_x,unsigned int *gridDim_y,unsigned int *blockDim_x_ascii,unsigned int *blockDim_y_ascii,
                                unsigned int *gridDim_x_ascii,unsigned int *gridDim_y_ascii,unsigned int *nb_sleep_thread_x,unsigned int *nb_sleep_thread_y,unsigned int *nb_sleep_thread_x_ascii,
                                unsigned int *nb_sleep_thread_y_ascii,float argv1,long unsigned int *sz_in_bytes_img_ascii,float **img_ascii,float **d_img_ascii,struct lib_ascii ascii,unsigned int *blockDim_x_color, 
                                unsigned int *blockDim_y_color,unsigned int *gridDim_x_color,unsigned int *gridDim_y_color,unsigned int *nb_sleep_thread_x_color,unsigned int *nb_sleep_thread_y_color,
                                unsigned long *sz_in_bytes_ascii_color,unsigned int *width_color,unsigned int *height_color,unsigned char **img_ascii_color_final,unsigned char **d_img_ascii_color_final,FIBITMAP **bitmap_final,unsigned int *pitch_final)
{
  FreeImage_Initialise();
  
  char PathName_data_prep[100] = "images/frame0.jpg";
  FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName_data_prep);
  *bitmap = FreeImage_Load(FIF_JPEG, PathName_data_prep, 0);

  declaration_1(*bitmap,width,height,pitch);
  *sz_in_bytes = sizeof(unsigned int) * 3 * (*width) * (*height); //nb de valeurs pour toute image
  *img = (unsigned int*) malloc(*sz_in_bytes);
  hipMalloc((void**)&*d_img, *sz_in_bytes);

  declaration_2(*width,*height,blockDim_x,blockDim_y,gridDim_x,gridDim_y,blockDim_x_ascii,blockDim_y_ascii,gridDim_x_ascii,gridDim_y_ascii,
               nb_sleep_thread_x,nb_sleep_thread_y,nb_sleep_thread_x_ascii,nb_sleep_thread_y_ascii,argv1);
  *sz_in_bytes_img_ascii = sizeof(float)*(*gridDim_x_ascii)*(*gridDim_y_ascii);
  *img_ascii = (float*) malloc(*sz_in_bytes_img_ascii);
  hipMalloc((void**)&*d_img_ascii, 4*(*sz_in_bytes_img_ascii));

  *blockDim_x_color = ascii.kwidthCaracter; // 7 
  *blockDim_y_color = ascii.kheightCaracter; // 11
  *nb_sleep_thread_x_color = *nb_sleep_thread_x_ascii;
  *nb_sleep_thread_y_color = *nb_sleep_thread_y_ascii;
  declaration_3(*blockDim_x_color,*blockDim_y_color,gridDim_x_color,gridDim_y_color,*gridDim_x_ascii,*gridDim_y_ascii,sz_in_bytes_ascii_color,
                width_color,height_color);

  *img_ascii_color_final = (unsigned char*) malloc(*sz_in_bytes_ascii_color);
  hipMalloc((void **)&*d_img_ascii_color_final, *sz_in_bytes_ascii_color); 
  *bitmap_final = FreeImage_Allocate(*width_color,*height_color, 24);
  *pitch_final = FreeImage_GetPitch(*bitmap_final);

  FreeImage_DeInitialise();

}



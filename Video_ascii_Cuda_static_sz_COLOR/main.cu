#include <iostream>
#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <sys/time.h>
#include "FreeImage.h"
#include "lib_ascii.h"
#include "host.cu"
#include "device.cu"

#define BPP 24

// Assemblage video python //
static char decoupe[100] = "python3 decoupe_vid.py ";                                     // appel focntion python pour découper la vidéo
static char nbr_img[100] = {0};                                                           // nombre d'images dans la video
static char assemble[100] = "python3 assemble_vid.py ";                                   // appel focntion python pour assembler la vidéo
// Mesures temps //
static double start,stop,start_kernel,stop_kernel,
temps_reor,temps_ascii,temps_cut,temps_ass,
temps_kernel1_moyen,temps_kernel2_moyen;                                                  // variables pour mesurer le temps
static double *temps_kernel1,*temps_kernel2;                                              // pointeur tableau qui stock le temps des kernels à chaque itérations
static char num[10];                                                                      // pour le numéro des iamges (framex.png)
// Barre de chargement //
static char barre[200] = "Traitement ascii des images";                                   //
static float eps;                                                                         // pourcentage équivalent à 1 '#' dans la barre de chargement
static int taille;                                                                        //
static int max_it;                                                                      //
// Pour le traitement d'images //
static unsigned int blockDim_x;                                                           //
static unsigned int blockDim_y;                                                           //
static unsigned int gridDim_x;                                                            //
static unsigned int gridDim_y;                                                            // 
static unsigned int nb_characters;                                                        //
static unsigned int width;                                                                //
static unsigned int height;                                                               //
static unsigned int width_color;                                                          //
static unsigned int height_color;                                                         //                                                             
static unsigned int pitch;                                                                //
static unsigned int pitch_final;   
static long unsigned int sz_in_bytes;                                                     // nb de bytes pour l'image initiale
static long unsigned int sz_in_bytes_img_ascii;                                           // nb de bytes pour l'image moyennée
static long unsigned int sz_in_bytes_ascii_color;                                         //

static unsigned int *img;                                                                 // pointeur
static float *img_ascii;                                                                  // pointeur
static unsigned char *img_ascii_color_final;                                              //
static char *final_ascii;                                                                 // pointeur
static unsigned int *d_img;                                                               // pointeur
static const char *use_ascii;                                                             // pointeur du tableau des ascii qui seront utilisés dans l'image (trié par ordre de niveau de gris croissant)
static unsigned char *d_tab_ascii_lib;                                                    //
static float *d_img_ascii;                                                                       // pointeur
static unsigned char *d_img_ascii_color_final;                                            //

static unsigned int gridDim_x_ascii;                                                      // largeur de l'image en nombre d'ascii
static unsigned int gridDim_y_ascii;                                                      //
static unsigned int blockDim_x_ascii;                                                     //
static unsigned int blockDim_y_ascii;                                                     //
static unsigned int nb_sleep_thread_x;                                                    //
static unsigned int nb_sleep_thread_y;                                                    //
static unsigned int nb_sleep_thread_x_ascii;                                              //
static unsigned int nb_sleep_thread_y_ascii;                                              //
static unsigned int nb_sleep_thread_x_color;                                              //
static unsigned int nb_sleep_thread_y_color;                                              //

static unsigned int blockDim_x_color;                                                     //
static unsigned int blockDim_y_color;                                                     //
static unsigned int gridDim_x_color;                                                      //
static unsigned int gridDim_y_color;                                                      //

FIBITMAP *bitmap_final;
FIBITMAP *bitmap;
FREE_IMAGE_FORMAT fif;

float *temp_img;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main (int argc , char** argv)
{

   // ARG_ERROR(argc,argv[1],argv[2]);

   printf("\n[------------------ Réordonnancement du tableau ASCII utilisé pour générer l'image (ordre croissant en niveau de gris) ------------------]\n\n");
   start = get_time();
   // ############ Librairie ascii ############
   use_ascii = "8$&03421*! "; // 11
   nb_characters = strlen(use_ascii);
   lib_ascii ascii{use_ascii, use_ascii + nb_characters};
   affiche_ascii(ascii);
   hipMalloc((void**)&d_tab_ascii_lib,ascii.RawSize());
   hipMemcpy(d_tab_ascii_lib,ascii.CaracterArray(),ascii.RawSize(),hipMemcpyHostToDevice);
   gpuErrchk( hipPeekAtLastError() );
   stop = get_time();
   temps_reor = stop-start;
   printf("\nTemps réordonnancement video : %f secondes\n",temps_reor);
   // #########################################

   printf("\n[------------------ DECOUPAGE DE LA VIDEO ------------------]\n");
   start = get_time();
   strcat(decoupe, argv[4]);
   system(decoupe);
   stop = get_time();
   temps_cut = stop-start;
   printf("\nTemps découpage video : %f secondes \n\n",temps_cut);

   printf("[------------------ TRAITEMENT ASCII DES IMAGES ------------------]\n\n");
   system("rm -r images_ascii/");
   system("mkdir images_ascii");

   FILE * f_img = popen("find images -type f | wc -l","r");
   fgets(nbr_img, 100, f_img); // calcul le nombre d'image à transformer
   pclose(f_img);

   // Pour la barre de chargement
   init_barre_chargement(barre,&taille,&eps,&max_it,nbr_img);

   temps_kernel1 = (double*) malloc(max_it*sizeof(double));
   temps_kernel2 = (double*) malloc(max_it*sizeof(double));
   data_preparation(&bitmap,&width,&height,&pitch,&sz_in_bytes,&img,&d_img,&blockDim_x,&blockDim_y,&gridDim_x,&gridDim_y,&blockDim_x_ascii,&blockDim_y_ascii,
                    &gridDim_x_ascii,&gridDim_y_ascii,&nb_sleep_thread_x,&nb_sleep_thread_y,&nb_sleep_thread_x_ascii,&nb_sleep_thread_y_ascii,atof(argv[1]),
                    &sz_in_bytes_img_ascii,&img_ascii,&d_img_ascii,ascii,&blockDim_x_color, &blockDim_y_color,&gridDim_x_color,&gridDim_y_color,&nb_sleep_thread_x_color,
                    &nb_sleep_thread_y_color,&sz_in_bytes_ascii_color,&width_color,&height_color,&img_ascii_color_final,&d_img_ascii_color_final,&bitmap_final,&pitch_final);

   FreeImage_Initialise();
   start = get_time();
   for(int k=0; k<max_it;k++){

      barre_chargement(barre,100*(k+1)/max_it,k+1,max_it,eps,taille);
      char PathName[100] = "images/frame";
      sprintf(num, "%d", k);
      strcat(PathName, num);
      strcat(PathName,".jpg");

      bitmap = FreeImage_Load(FIF_JPEG, PathName, 0);

      REORDER_IMG(img,height,width,pitch,bitmap);

      hipMemcpy(d_img, img, sz_in_bytes,hipMemcpyHostToDevice);
      gpuErrchk( hipPeekAtLastError() );

      hipMemset(d_img_ascii,0.,4*sz_in_bytes_img_ascii);                                                
      gpuErrchk( hipPeekAtLastError() );

      dim3 dimBlock(blockDim_x,blockDim_y,1);
      dim3 dimGrid(gridDim_x,gridDim_y,1);
      start_kernel = get_time();
      Niveau_Gris_Color_Moyennage<<<dimGrid, dimBlock>>>(d_img_ascii,d_img,width,nb_sleep_thread_x,nb_sleep_thread_y_ascii,
                                                         gridDim_x_ascii,blockDim_x_ascii,blockDim_y_ascii);
      stop_kernel = get_time();
      temps_kernel1[k] = stop_kernel-start_kernel;
      gpuErrchk( hipPeekAtLastError() );
    
      dim3 dimBlock_color(blockDim_x_color,blockDim_y_color,1);
      dim3 dimGrid_color(gridDim_x_color,gridDim_y_color,1);   
      
      start_kernel = get_time();
      Image_Color<<<dimGrid_color, dimBlock_color>>>(d_img_ascii,d_img_ascii_color_final,d_tab_ascii_lib,width_color,
                                                     ascii.kwidthCaracter,ascii.kheightCaracter,nb_characters,gridDim_x_ascii,
                                                     gridDim_y_ascii,blockDim_x_ascii,blockDim_y_ascii,nb_sleep_thread_x_color,nb_sleep_thread_y_color,
                                                     atoi(argv[2]),atoi(argv[3]));
      stop_kernel = get_time();
      temps_kernel2[k] = stop_kernel-start_kernel; 
      gpuErrchk( hipPeekAtLastError() ); 
      
      hipMemcpy(img_ascii_color_final, d_img_ascii_color_final, sz_in_bytes_ascii_color, hipMemcpyDeviceToHost);   
      gpuErrchk( hipPeekAtLastError() );

      SAVE_IMG(img_ascii_color_final,height_color,width_color,pitch_final,bitmap_final,k);  // Créer le pitch&bitmap de img_ascii_color
   }
   FreeImage_DeInitialise();
   stop = get_time();
   temps_ascii = stop-start;
   free(img);
   free(final_ascii);
   free(img_ascii);
   hipFreeAsync(d_tab_ascii_lib,0);
   hipFreeAsync(d_img_ascii,0);
   hipFreeAsync(d_img_ascii_color_final,0);

   printf("\nTemps moyen traitement ascii : %f secondes\n",temps_ascii/max_it);
   temps_kernel1_moyen = 0, temps_kernel2_moyen = 0;
   for(int k=0; k<max_it;k++){
      temps_kernel1_moyen += temps_kernel1[k];
      temps_kernel2_moyen += temps_kernel2[k];
   }
   printf("Temps moyen pour le traitement du kernel \"Niveau_Gris_Color_Moyennage\": %f secondes\n",temps_kernel1_moyen/max_it);
   printf("Temps moyen pour le traitement du kernel \"Image_Color\" : %f secondes\n",temps_kernel2_moyen/max_it);
   printf("\nTemps total traitement ascii : %f secondes\n\n",temps_ascii);

   start = get_time();
   printf("[------------------ ASSEMBLAGE DE LA VIDEO ------------------]\n\n");
   strcat(assemble, argv[4]);
   system(assemble);
   stop = get_time();
   temps_ass = stop-start;

   printf("\nVideo bien assemblée.\n\nTemps assemblage video : %f secondes\n\n",stop-start);
   printf("----------------------------------\n");
   printf("TEMPS TOTAL = %f secondes \n",temps_reor+temps_cut+temps_ascii+temps_ass);
   printf("----------------------------------\n\n");
   return 0;
}

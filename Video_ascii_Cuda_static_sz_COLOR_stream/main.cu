#include <iostream>
#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <sys/time.h>
#include "FreeImage.h"
#include "lib_ascii.h"
#include "host.cu"
#include "device.cu"

#define BPP 24                                                                            // Bytes Per Pixels (24 => 3*8)
#define NB_STREAMS 4                                                                      // NB_STREAMS > 1

static int nb_streams=NB_STREAMS;
// Assemblage video python //
static char decoupe[100] = "python3 decoupe_vid.py ";                                     // appel focntion python pour découper la vidéo
static char nbr_img[100] = {0};                                                           // nombre d'images dans la video
static char assemble[100] = "python3 assemble_vid.py ";                                   // appel focntion python pour assembler la vidéo
// Mesures temps //
static double start,stop,start_kernel,stop_kernel,
temps_reor,temps_ascii,temps_cut,temps_ass,
temps_kernel1_moyen,temps_kernel2_moyen;                                                  // variables pour mesurer le temps
static double *temps_kernel1,*temps_kernel2;                                              // pointeur tableau qui stock le temps des kernels à chaque itérations
static char PathName1[100],PathName2[100],PathName3[100],PathName4[100];
static char num[10];                                                                      // pour le numéro des iamges (framex.png)
// Barre de chargement //
static char barre[200] = "Traitement ascii des images";                                   //
static float eps;                                                                         // pourcentage équivalent à 1 '#' dans la barre de chargement
static int taille;                                                                        //
static int max_it;                                                                        //
// Pour le traitement d'images //
static unsigned int blockDim_x;                                                           //
static unsigned int blockDim_y;                                                           //
static unsigned int gridDim_x;                                                            //
static unsigned int gridDim_y;                                                            // 
static unsigned int nb_characters;                                                        //
static unsigned int width;                                                                //
static unsigned int height;                                                               //
static unsigned int width_color;                                                          //
static unsigned int height_color;                                                         //                                                             
static unsigned int pitch;                                                                //
static long unsigned int sz_in_bytes;                                                     // nb de bytes pour l'image initiale
static long unsigned int sz_in_bytes_img_ascii;                                           // nb de bytes pour l'image moyennée
static long unsigned int sz_in_bytes_ascii_color;                                         //

static unsigned int *img1,*img2,*img3,*img4;                                              // pointeur
static float *img_ascii1,*img_ascii2,*img_ascii3,*img_ascii4;                             // pointeur
static unsigned char *img_ascii_color_final1,*img_ascii_color_final2,                     //
                     *img_ascii_color_final3,*img_ascii_color_final4;                     //
static char *final_ascii1,*final_ascii2,*final_ascii3,*final_ascii4;                      // pointeur
static unsigned int *d_img1,*d_img2,*d_img3,*d_img4;                                      // pointeur
static const char *use_ascii;                                                             // pointeur du tableau des ascii qui seront utilisés dans l'image (trié par ordre de niveau de gris croissant)
static unsigned char *d_tab_ascii_lib;                                                    //
static float *d_img_ascii1,*d_img_ascii2,*d_img_ascii3,*d_img_ascii4;                     // pointeur
static unsigned char *d_img_ascii_color_final1,*d_img_ascii_color_final2,
                     *d_img_ascii_color_final3,*d_img_ascii_color_final4;                 //

static unsigned int gridDim_x_ascii;                                                      // largeur de l'image en nombre d'ascii
static unsigned int gridDim_y_ascii;                                                      //
static unsigned int blockDim_x_ascii;                                                     //
static unsigned int blockDim_y_ascii;                                                     //
static unsigned int nb_sleep_thread_x;                                                    //
static unsigned int nb_sleep_thread_y;                                                    //
static unsigned int nb_sleep_thread_x_ascii;                                              //
static unsigned int nb_sleep_thread_y_ascii;                                              //
static unsigned int nb_sleep_thread_x_color;                                              //
static unsigned int nb_sleep_thread_y_color;                                              //

static unsigned int blockDim_x_color;                                                     //
static unsigned int blockDim_y_color;                                                     //
static unsigned int gridDim_x_color;                                                      //
static unsigned int gridDim_y_color;                                                      //

FIBITMAP *bitmap1_final,*bitmap2_final,*bitmap3_final,*bitmap4_final;
FIBITMAP *bitmap1,*bitmap2,*bitmap3,*bitmap4;
FREE_IMAGE_FORMAT fif;

static hipStream_t stream[NB_STREAMS];                                                   //

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main (int argc , char** argv)
{

   // ARG_ERROR(argc,argv[1],argv[2]);

   printf("\n[------------------ Réordonnancement du tableau ASCII utilisé pour générer l'image (ordre croissant en niveau de gris) ------------------]\n\n");
   start = get_time();
   // ############ Librairie ascii ############
   use_ascii = "8$&03421*! "; // 11
   nb_characters = strlen(use_ascii);
   lib_ascii ascii{use_ascii, use_ascii + nb_characters};
   affiche_ascii(ascii);
   hipMalloc((void**)&d_tab_ascii_lib,ascii.RawSize());
   hipMemcpy(d_tab_ascii_lib,ascii.CaracterArray(),ascii.RawSize(),hipMemcpyHostToDevice);
   gpuErrchk( hipPeekAtLastError() );
   stop = get_time();
   temps_reor = stop-start;
   printf("\nTemps réordonnancement video : %f secondes\n",temps_reor);
   // #########################################

   printf("\n[------------------ DECOUPAGE DE LA VIDEO ------------------]\n");
   start = get_time();
   strcat(decoupe, argv[4]);
   system(decoupe);
   stop = get_time();
   temps_cut = stop-start;
   printf("\nTemps découpage video : %f secondes \n\n",temps_cut);

   printf("[------------------ TRAITEMENT ASCII DES IMAGES ------------------]\n\n");
   system("rm -r images_ascii/");
   system("mkdir images_ascii");

   FILE * f_img = popen("find images -type f | wc -l","r");
   fgets(nbr_img, 100, f_img); // calcul le nombre d'image à transformer
   pclose(f_img);
   // Pour la barre de chargement
   init_barre_chargement(barre,&taille,&eps,&max_it,nbr_img);

   temps_kernel1 = (double*) malloc(max_it*sizeof(double));
   temps_kernel2 = (double*) malloc(max_it*sizeof(double));
   data_preparation(&bitmap1,&width,&height,&pitch,&sz_in_bytes,&img1,&img2,&img3,&img4,&d_img1,&d_img2,&d_img3,&d_img4,&blockDim_x,&blockDim_y,&gridDim_x,
                    &gridDim_y,&blockDim_x_ascii,&blockDim_y_ascii,&gridDim_x_ascii,&gridDim_y_ascii,&nb_sleep_thread_x,&nb_sleep_thread_y,&nb_sleep_thread_x_ascii,
                    &nb_sleep_thread_y_ascii,atof(argv[1]),&sz_in_bytes_img_ascii,
                    &img_ascii1,&img_ascii2,&img_ascii3,&img_ascii4,
                    &d_img_ascii1,&d_img_ascii2,&d_img_ascii3,&d_img_ascii4,ascii,
                    &blockDim_x_color, &blockDim_y_color,&gridDim_x_color,&gridDim_y_color,&nb_sleep_thread_x_color,
                    &nb_sleep_thread_y_color,&sz_in_bytes_ascii_color,&width_color,&height_color,&img_ascii_color_final1,&img_ascii_color_final2,&img_ascii_color_final3,&img_ascii_color_final4,
                    &d_img_ascii_color_final1,&d_img_ascii_color_final2,&d_img_ascii_color_final3,&d_img_ascii_color_final4,
                    &bitmap1_final,&bitmap2_final,&bitmap3_final,&bitmap4_final);

   FreeImage_Initialise();
   start = get_time();

   for(int k=0; k<max_it;k=k+NB_STREAMS){

      // printf("\nmax_it = %d, k0 = %d, k1 = %d, k2 = %d, k3 = %d \n",max_it,k-nb_streams+0,k-nb_streams+1,k-nb_streams+2,k-nb_streams+3);

      START_IT(barre,k,max_it,eps,taille,NB_STREAMS,&nb_streams,PathName1,PathName2,PathName3,PathName4,num);

      // printf("\nk0 = %d, k1 = %d, k2 = %d, k3 = %d \n",k-nb_streams+0,k-nb_streams+1,k-nb_streams+2,k-nb_streams+3);


      bitmap1 = FreeImage_Load(FIF_JPEG, PathName1, 0);
      bitmap2 = FreeImage_Load(FIF_JPEG, PathName2, 0);
      bitmap3 = FreeImage_Load(FIF_JPEG, PathName3, 0);
      bitmap4 = FreeImage_Load(FIF_JPEG, PathName4, 0);

      REORDER_IMG(img1,height,width,pitch,bitmap1);
      REORDER_IMG(img2,height,width,pitch,bitmap2);
      REORDER_IMG(img3,height,width,pitch,bitmap3);
      REORDER_IMG(img4,height,width,pitch,bitmap4);

      hipStreamCreate(&stream[0]);
      hipStreamCreate(&stream[1]);
      hipStreamCreate(&stream[2]);
      hipStreamCreate(&stream[3]);

      hipMemcpyAsync(d_img1, img1, sz_in_bytes,hipMemcpyHostToDevice,stream[0]);
      hipMemcpyAsync(d_img2, img2, sz_in_bytes,hipMemcpyHostToDevice,stream[1]);
      hipMemcpyAsync(d_img3, img3, sz_in_bytes,hipMemcpyHostToDevice,stream[2]);
      hipMemcpyAsync(d_img4, img4, sz_in_bytes,hipMemcpyHostToDevice,stream[3]);
      gpuErrchk( hipPeekAtLastError() );

      hipMemsetAsync(d_img_ascii1,0.,4*sz_in_bytes_img_ascii,stream[0]);      
      hipMemsetAsync(d_img_ascii2,0.,4*sz_in_bytes_img_ascii,stream[1]);  
      hipMemsetAsync(d_img_ascii3,0.,4*sz_in_bytes_img_ascii,stream[2]);  
      hipMemsetAsync(d_img_ascii4,0.,4*sz_in_bytes_img_ascii,stream[3]);                                            
      gpuErrchk( hipPeekAtLastError() );


      //############################### FIRST KERNEL ######################################  
      dim3 dimBlock(blockDim_x,blockDim_y,1);
      dim3 dimGrid(gridDim_x,gridDim_y,1);
      // start_kernel = get_time();
      Niveau_Gris_Color_Moyennage<<<dimGrid, dimBlock, 0, stream[0]>>>(d_img_ascii1,d_img1,width,nb_sleep_thread_x,nb_sleep_thread_y_ascii,gridDim_x_ascii,blockDim_x_ascii,blockDim_y_ascii);
      Niveau_Gris_Color_Moyennage<<<dimGrid, dimBlock, 0, stream[1]>>>(d_img_ascii2,d_img2,width,nb_sleep_thread_x,nb_sleep_thread_y_ascii,gridDim_x_ascii,blockDim_x_ascii,blockDim_y_ascii);
      Niveau_Gris_Color_Moyennage<<<dimGrid, dimBlock, 0, stream[2]>>>(d_img_ascii3,d_img3,width,nb_sleep_thread_x,nb_sleep_thread_y_ascii,gridDim_x_ascii,blockDim_x_ascii,blockDim_y_ascii);
      Niveau_Gris_Color_Moyennage<<<dimGrid, dimBlock, 0, stream[3]>>>(d_img_ascii4,d_img4,width,nb_sleep_thread_x,nb_sleep_thread_y_ascii,gridDim_x_ascii,blockDim_x_ascii,blockDim_y_ascii);
      // stop_kernel = get_time();
      // temps_kernel1[k] = stop_kernel-start_kernel;
      gpuErrchk( hipPeekAtLastError() );
      //############################ END FIRST KERNEL ###################################### 
    
      //############################## SECOND KERNEL #######################################   
      dim3 dimBlock_color(blockDim_x_color,blockDim_y_color,1);
      dim3 dimGrid_color(gridDim_x_color,gridDim_y_color,1);   
      
      // start_kernel = get_time();
      Image_Color<<<dimGrid_color, dimBlock_color ,0 ,stream[0]>>>(d_img_ascii1,d_img_ascii_color_final1,d_tab_ascii_lib,width_color,ascii.kwidthCaracter,ascii.kheightCaracter,nb_characters,gridDim_x_ascii,
                                                                   gridDim_y_ascii,blockDim_x_ascii,blockDim_y_ascii,nb_sleep_thread_x_color,nb_sleep_thread_y_color,atoi(argv[2]),atoi(argv[3]));
      Image_Color<<<dimGrid_color, dimBlock_color ,0 ,stream[1]>>>(d_img_ascii2,d_img_ascii_color_final2,d_tab_ascii_lib,width_color,ascii.kwidthCaracter,ascii.kheightCaracter,nb_characters,gridDim_x_ascii,
                                                                   gridDim_y_ascii,blockDim_x_ascii,blockDim_y_ascii,nb_sleep_thread_x_color,nb_sleep_thread_y_color,atoi(argv[2]),atoi(argv[3]));
      Image_Color<<<dimGrid_color, dimBlock_color ,0 ,stream[2]>>>(d_img_ascii3,d_img_ascii_color_final3,d_tab_ascii_lib,width_color,ascii.kwidthCaracter,ascii.kheightCaracter,nb_characters,gridDim_x_ascii,
                                                                   gridDim_y_ascii,blockDim_x_ascii,blockDim_y_ascii,nb_sleep_thread_x_color,nb_sleep_thread_y_color,atoi(argv[2]),atoi(argv[3]));
      Image_Color<<<dimGrid_color, dimBlock_color ,0 ,stream[3]>>>(d_img_ascii4,d_img_ascii_color_final4,d_tab_ascii_lib,width_color,ascii.kwidthCaracter,ascii.kheightCaracter,nb_characters,gridDim_x_ascii,
                                                                   gridDim_y_ascii,blockDim_x_ascii,blockDim_y_ascii,nb_sleep_thread_x_color,nb_sleep_thread_y_color,atoi(argv[2]),atoi(argv[3]));      
      // stop_kernel = get_time();
      // temps_kernel2[k] = stop_kernel-start_kernel; 
      gpuErrchk( hipPeekAtLastError() ); 
      //############################ END SECOND KERNEL ######################################
      
      hipMemcpyAsync(img_ascii_color_final1, d_img_ascii_color_final1, sz_in_bytes_ascii_color, hipMemcpyDeviceToHost,stream[0]);   
      hipMemcpyAsync(img_ascii_color_final2, d_img_ascii_color_final2, sz_in_bytes_ascii_color, hipMemcpyDeviceToHost,stream[1]);  
      hipMemcpyAsync(img_ascii_color_final3, d_img_ascii_color_final3, sz_in_bytes_ascii_color, hipMemcpyDeviceToHost,stream[2]);  
      hipMemcpyAsync(img_ascii_color_final4, d_img_ascii_color_final4, sz_in_bytes_ascii_color, hipMemcpyDeviceToHost,stream[3]);  
      gpuErrchk( hipPeekAtLastError() );

      SAVE_IMG(img_ascii_color_final1,height_color,width_color,bitmap1_final,k-nb_streams+0);  // Créer le pitch&bitmap de img_ascii_color
      SAVE_IMG(img_ascii_color_final2,height_color,width_color,bitmap2_final,k-nb_streams+1);  // Créer le pitch&bitmap de img_ascii_color
      SAVE_IMG(img_ascii_color_final3,height_color,width_color,bitmap3_final,k-nb_streams+2);  // Créer le pitch&bitmap de img_ascii_color
      SAVE_IMG(img_ascii_color_final4,height_color,width_color,bitmap4_final,k-nb_streams+3);  // Créer le pitch&bitmap de img_ascii_color

      hipStreamDestroy(stream[0]);
      hipStreamDestroy(stream[1]);
      hipStreamDestroy(stream[2]);
      hipStreamDestroy(stream[3]);
   }
   FreeImage_DeInitialise();
   stop = get_time();
   temps_ascii = stop-start;
   free(img1);
   free(img2);
   free(img3);
   free(img4);
   free(final_ascii1);
   free(final_ascii2);
   free(final_ascii3);
   free(final_ascii4);
   free(img_ascii1);
   free(img_ascii2);
   free(img_ascii3);
   free(img_ascii4);
   free(img_ascii_color_final1);
   free(img_ascii_color_final2);
   free(img_ascii_color_final3);
   free(img_ascii_color_final4);
   hipFreeAsync(d_tab_ascii_lib,0);
   hipFreeAsync(d_img_ascii1,0);
   hipFreeAsync(d_img_ascii2,0);
   hipFreeAsync(d_img_ascii3,0);
   hipFreeAsync(d_img_ascii4,0);
   hipFreeAsync(d_img_ascii_color_final1,0);
   hipFreeAsync(d_img_ascii_color_final2,0);
   hipFreeAsync(d_img_ascii_color_final3,0);
   hipFreeAsync(d_img_ascii_color_final4,0);

   printf("\nTemps moyen traitement ascii : %f secondes\n",temps_ascii/max_it);
   // temps_kernel1_moyen = 0, temps_kernel2_moyen = 0;
   // for(int k=0; k<max_it;k++){
   //    temps_kernel1_moyen += temps_kernel1[k];
   //    temps_kernel2_moyen += temps_kernel2[k];
   // }
   // printf("Temps moyen pour le traitement du kernel \"Niveau_Gris_Color_Moyennage\": %f secondes\n",temps_kernel1_moyen/max_it);
   // printf("Temps moyen pour le traitement du kernel \"Image_Color\" : %f secondes\n",temps_kernel2_moyen/max_it);
   printf("\nTemps total traitement ascii : %f secondes\n\n",temps_ascii);

   start = get_time();
   printf("[------------------ ASSEMBLAGE DE LA VIDEO ------------------]\n\n");
   strcat(assemble, argv[4]);
   system(assemble);
   stop = get_time();
   temps_ass = stop-start;

   printf("\nVideo bien assemblée.\n\nTemps assemblage video : %f secondes\n\n",stop-start);
   printf("----------------------------------\n");
   printf("TEMPS TOTAL = %f secondes \n",temps_reor+temps_cut+temps_ascii+temps_ass);
   printf("----------------------------------\n\n");
   return 0;
}

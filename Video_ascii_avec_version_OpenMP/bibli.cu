
#include <hip/hip_runtime.h>

double start,stop,start_kernel,stop_kernel,cpu_time_used,temps_kernel_moyen;
double *temps_kernel;
char num[10]; // Pour le num de framex.png

// Pour la barre de chargement
float p_avant = 0.;
float eps = 1.5; // pourcentage pour 1 '#' dans la barre
char barre[200];


unsigned width;
unsigned height;
unsigned pitch;


int sz_in_bytes;

unsigned int *img;

unsigned int *d_img;

// Pour le prétraitement d'images
int block_x;
int grid_x;
int block_y;
int grid_y;


// Pour le traitement ascii
int DETAIL; //nombre d'ascii différents

int grid_x_ascii; // largeur de l'image en nombre d'ascii
int grid_y_ascii;

int block_x_ascii;
int block_y_ascii;

int nb_sleep_thread_x_all; // nombres de threads inatif selon x
int nb_sleep_thread_y_all; // nombres de threads inatif selon y
int nb_sleep_block_x; // nombre de blocks inactifs selon x (arrondie inférieur)
int nb_sleep_block_y; // nombre de blocs inactis selon y (arrondie inférieur)



float *img_ascii;
float *d_img_ascii;

int MAX;

char *final_ascii;

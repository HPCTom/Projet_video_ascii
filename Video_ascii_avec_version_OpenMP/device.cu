
#include <hip/hip_runtime.h>
//##############################################################################################
//################################### FONCTION DEVICE ##########################################
//##############################################################################################


__global__ void Niveau_Gris(unsigned int *d_img, unsigned width, unsigned height) // NIVEAU DE GRIS
{

	int n_x = gridDim.x*blockDim.x; // nombre de threads par ligne
	int n_y = gridDim.y*blockDim.y; // nombres de threads par colonne

	int nb_sleep_thread_x = n_x-width; // nombres de threads inatif par ligne
	int nb_sleep_thread_y = n_y-height; // nombre de threads inactif par colonne

	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){
			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			int idx = ((y * width) + x) * 3;

			d_img[idx+0] = 0.299*d_img[idx+0]+0.587*d_img[idx+1]+0.114*d_img[idx+2];
			d_img[idx+1] = d_img[idx+0];
			d_img[idx+2] = d_img[idx+0];
		}
	}
}


__global__ void Moyennage(float *d_img_ascii, unsigned int* d_tmp, unsigned width,unsigned height)
{

	int n_x = gridDim.x*blockDim.x; // nombre de threads par ligne
	int n_y = gridDim.y*blockDim.y; // nombres de threads par colonne

	int nb_sleep_thread_x = n_x-width; // nombres de threads inatif par ligne
	int nb_sleep_thread_y = n_y-height; // nombre de threads inactif par colonne

	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){

			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			int idx = ((y * width) + width - x) * 3;
			int idx_ascii  = gridDim.x*blockIdx.y + blockIdx.x;

			atomicAdd(&d_img_ascii[idx_ascii],(float)d_tmp[idx]);

		}
	}

}


__global__ void Niveau_Gris_Moyennage(float *d_img_ascii, unsigned int* d_img, unsigned width,unsigned height)
{

	int n_x = gridDim.x*blockDim.x; // nombre de threads par ligne
	int n_y = gridDim.y*blockDim.y; // nombres de threads par colonne

	int nb_sleep_thread_x = n_x-width; // nombres de threads inatif par ligne
	int nb_sleep_thread_y = n_y-height; // nombre de threads inactif par colonne

	if(blockIdx.y < gridDim.y-1 || blockIdx.y == gridDim.y-1 && threadIdx.y < blockDim.y-nb_sleep_thread_y){
		if(blockIdx.x < gridDim.x-1 || blockIdx.x == gridDim.x-1 && threadIdx.x < blockDim.x-nb_sleep_thread_x){

			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			int idx = ((y * width) + width - x) * 3;
			int idx_ascii  = gridDim.x*blockIdx.y + blockIdx.x;

			atomicAdd(&d_img_ascii[idx_ascii],0.299*d_img[idx+0]+0.587*d_img[idx+1]+0.114*d_img[idx+2]);

		}
	}

}

#include "hip/hip_runtime.h"
//##############################################################################################
//################################### FONCTION HOST ############################################
//##############################################################################################

//#### Gestion des erreurs dans les parametres d'entré #####
__host__ void error_msg(int argc,int argv1, int argv2, int argv3,int argv4,int width){
	if(argc != 6 && argc != 7){
		printf("\nUsage : le programme prend 6 arguments.\n"
					 "argv[1] = le nombre de threads par bloc selon x.\n"
					 "argv[2] = le nombre de threads par bloc selon y.\n"
					 "argv[3] = pourcentage de résolution de l'image entre 0 et 100 (100 %% indique qu'il y aura autant d'ascii en largeur que de pixels).\n"
					 "argv[4] = nombre d'ascii différents utilisés pour générer l'image.\n"
           "argv[5] = nom de la video.\n"
           "Exemple: ./modif_img 30 30 100 10 \n\n");
	}

  if(argv1*argv2 > 1024){
    printf("\nNombre de threads par bloc possible dépassé, argv[1]*argv[2] doit etre inférieur à 1024. \n\n");
  }
	if(argv3 > 100 || argv3 <= 0){
		printf("\nLe pourcentage de résolution de l'image doit etre compris entre 0 (exclu) et 100 (cf README).\n\n");
	}
}

__host__ void ratio(float fact, int width, int height,int largeur_ascii,int *hauteur_ascii){
	// fact est le facteur qui compense l'ecart des caractere entre ses voisin gauche/droite et haut/bas
	float ratio = (float)width/(float)height*fact;
	*hauteur_ascii = largeur_ascii/ratio;

}

//#### Declaration des parametres
__host__ void declaration_1(FIBITMAP *bitmap,unsigned *width,unsigned *height,unsigned *pitch,unsigned int *img,int argc,int argv1, int argv2, int argv3, int argv4){
    *width  = FreeImage_GetWidth(bitmap);
    *height = FreeImage_GetHeight(bitmap);
    *pitch  = FreeImage_GetPitch(bitmap);
    error_msg(argc,argv1,argv2,argv3,argv4,*width); // gestion des erreur les arguments d'entrée
}

__host__ void declaration_2(int *block_x,int *block_y, int *grid_x, int *grid_y,int width, int height, int argv1, int argv2,int k){
  if(k==0){
    *block_x = argv1;
    *grid_x = ceil((float)width/(float)(*block_x));
    *block_y = argv2;
    *grid_y = ceil((float)height/(float)(*block_y));
  }
}

__host__ void declaration_3(int *DETAIL, int *grid_x_ascii,int *grid_y_ascii,int *block_x_ascii,int *block_y_ascii, float *img_ascii,int width, int height, float argv3, int argv4){
  *DETAIL = argv4; //nombre d'ascii différents

  *grid_x_ascii = (int)((float)width*argv3/100.); // largeur de l'image en nombre d'ascii

	//ratio(1.8,width,height,*grid_x_ascii,grid_y_ascii)

  float ratio = (float)width/(float)height*1.8; //calcul la hauteur de l'image en prennant en compte le ratio de l'image Hauteur/Largeur et compense le fait que les characteres soit plus espacés en hauteur que en largeur (1.8) dans un fihcier texte.
	*grid_y_ascii = *grid_x_ascii/ratio;

  *block_x_ascii = ceil((float)width/(float)(*grid_x_ascii));
  *block_y_ascii = ceil((float)height/(float)(*grid_y_ascii));

  int nb_sleep_thread_x_all = *grid_x_ascii*(*block_x_ascii)-width; // nombres de threads inactif selon x
  int nb_sleep_thread_y_all = *grid_y_ascii*(*block_y_ascii)-height; // nombres de threads inactif selon y
  int nb_sleep_block_x = nb_sleep_thread_x_all/(*block_x_ascii); // nombre de blocks inactifs selon x (arrondie inférieur)
  int nb_sleep_block_y = nb_sleep_thread_y_all/(*block_y_ascii); // nombre de blocs inactis selon y (arrondie inférieur)

  *grid_x_ascii = *grid_x_ascii - nb_sleep_block_x; // nouvelle grid en x redimensionnée
  *grid_y_ascii = *grid_y_ascii - nb_sleep_block_y; // nouvelle grid en y redimensionnée

}


void init_barre_chargement(char *barre,int *cpt,float eps,int max_it){
  strcat(barre, " [");
  while(barre[*cpt]!='\0'){
    *cpt = *cpt+1;
  }
  int taille = (int)ceil(100./eps); //nombre de #

  for(int t=0;t<taille;t++){
    strcat(barre, " ");
  }
  strcat(barre, "]");
}

void barre_chargement(char *barre,float p,int k, float max, float eps,int taille){ //entier entre 0 et 100

  int idx = (int)(p/eps); // a quelle intervalle j'appartiens

  if(k==max){
    for(int k=taille; k<taille+idx+1; k++){
      barre[k] = '#';
    }
    printf("\r%s  %.2f%% \n",barre,p);
    fflush(stdout);
  }
  else{
    for(int k=taille; k<taille+idx; k++){
      barre[k] = '#';
    }
    printf("\r%s  %.2f%% ",barre,p);
    fflush(stdout);
  }
}
__host__ void barre_chargement_openmp(char *barre,float p,int k, int max, float eps,int taille,int max_it){ //entier entre 0 et 100
  int idx = (int)(p/eps); // a quelle intervalle jn'appartiens

  if(k==max-1 && max_it%max == 0){
    for(int k=taille; k<taille+idx+1; k++){
      barre[k] = '#';
    }
    printf("\r%s  100.00%% \n",barre);
    fflush(stdout);
  }

  else if(k==max){
    for(int k=taille; k<taille+idx; k++){
      barre[k] = '#';
    }
    printf("\r%s  100.00%% \n",barre);
    fflush(stdout);
  }
  else{
    for(int k=taille; k<taille+idx; k++){
      barre[k] = '#';
    }
    printf("\r%s  %.2f%% ",barre,p);
    fflush(stdout);
  }
}

//#### Permet de calculer le temps #####
__host__ double get_time() {
  struct timeval tv;
  gettimeofday(&tv, (void *)0);
  return (double) tv.tv_sec + tv.tv_usec*1e-6;
}

//#### Ordonne le tableau en RGB #####
__host__ void REORDER_IMG(unsigned int *img,int  height, int width,unsigned pitch,FIBITMAP* bitmap){
	BYTE *bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = bits;
    for ( int x =0; x<width; x++)
    {
      int idx = ((y * width) + x) * 3;
      img[idx + 0] = pixel[FI_RGBA_RED];
      img[idx + 1] = pixel[FI_RGBA_GREEN];
      img[idx + 2] = pixel[FI_RGBA_BLUE];
      pixel += 3;
    }
    bits += pitch;
  }
}

//#### Sauvergarde l'image finale #####
__host__ void SAVE_IMG(unsigned int *img,int  height, int width,const char *PathDest,unsigned pitch,FIBITMAP* bitmap){
	BYTE* bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = (BYTE*)bits;
    for ( int x =0; x<width; x++)
    {
      RGBQUAD newcolor;

      int idx = ((y * width) + x) * 3;
      newcolor.rgbRed = img[idx + 0];
      newcolor.rgbGreen = img[idx + 1];
      newcolor.rgbBlue = img[idx + 2];

      if(!FreeImage_SetPixelColor(bitmap, x, y, &newcolor))
      { fprintf(stderr, "(%d, %d) Fail...\n", x, y); }

      pixel+=3;
    }
    // next line
    bits += pitch;
  }

  if( FreeImage_Save (FIF_PNG, bitmap , PathDest , 0 ))
  FreeImage_DeInitialise(); //Cleanup !
}

__host__ int max_tab(unsigned int *img, int block_x, int block_y,int grid_x, int grid_y) //calcul le max du tableau en niveau de gris
{
	int taille = (grid_x)*(grid_y);
	int MAX = 0;
  for(int i=0;i<taille;i++)
  {
    if(img[i] > MAX){
      MAX=img[i];
    }
  }
	return MAX;
}

__host__ void choix_ascii(float *img_ascii,int taille,int taille_x,int taille_y,char *tab_final,
													int min, int MAX, int DETAIL){

	char ascii[255] = {'8','*','0','w','^','&','=','!','$','4','q','+','1','m','#','%','l',':','2','<','>','}','5','/','.','2','a','3','p','t','6','?','9','c','7','r','[',']','x','b'}; //40
  //char ascii[255] = {'8','&','4','w','^','*','=','!','$','0','q','+','1','m','#','%','l',':','2','<','>','}','5','/','.','2','a','3','p','t','6','?','9','c','7','r','[',']','x','b'}; //40

	if(DETAIL > MAX){
		printf("nombres d'ascii max dépacé DETAIL = %d et MAX = %d\n",DETAIL,MAX);
	}

	int eps = MAX/DETAIL;
	int moy;
	//printf("taille = %d, taille_x = %d, taille_y = %d\n",taille,taille_x,taille_y);

	for(int i=0;i<taille;i++)
	{
		moy = img_ascii[i]/((float)taille_x*(float)taille_y); //moyenne
		if(moy/eps > DETAIL-1){
			//printf("i = %d\n", i);
			tab_final[i] = ascii[moy/eps-1];
			//printf("indice = %d\n",moy/eps-1);


		}
		else{
			//printf("i2 = %d\n",i);
			tab_final[i] = ascii[moy/eps];
		}
	}

}

__host__ void tab_to_txt(char *final_ascii,float *img_ascii,char *tab,int hauteur_ascii,int largeur_ascii,
	 											 int taille_x, int taille_y,int MAX, int DETAIL)
{
	FILE *fp = NULL;
	fp = fopen(tab,"w");

	if(fp ==NULL)
	{
		printf("\ntab_to_txt : ERREUR OUVERTURE FICHIER\n");
	}

	choix_ascii(img_ascii,largeur_ascii*hauteur_ascii,taille_x,taille_y,final_ascii,0,MAX,DETAIL);

	int cpt = 0;

	for(int k=largeur_ascii*hauteur_ascii-1;k>-1;k--){

		if(cpt == largeur_ascii){
			cpt = 0;
			fprintf(fp,"\n");
		}
		cpt = cpt+1;
		if(k>0){
			fprintf(fp,"%c",final_ascii[k]);
		}
		else{
			fprintf(fp,"%c",final_ascii[0]);
		}
	}
	fclose(fp);

}

__host__ void txt_to_png(int width, int height,int largeur_ascii,char* tab_txt, char* tab_png,int MAX, int DETAIL)
{

	char ligne[width];

	sprintf(ligne,"convert -font Courier -background white -fill black label:@%s -flatten images_ascii/%s",tab_txt,tab_png);

  system(ligne);
}

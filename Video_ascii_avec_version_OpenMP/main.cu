#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <sys/time.h>
#include "FreeImage.h"
#include "bibli.cu"
#include "host.cu"
#include "device.cu"

int main (int argc , char** argv)
{

  printf("\n[------------------ DECOUPAGE DE LA VIDEO ------------------]\n");
  char decoupe[100] = "python3 decoupe_vid.py ";
  strcat(decoupe, argv[5]);
  system(decoupe);

  printf("[------------------ TRAITEMENT ASCII DES IMAGES ------------------]\n\n");
  system("rm -r images_ascii/");
  system("mkdir images_ascii");

  FILE * f_img = popen("find images -type f | wc -l","r");
  char nbr_img[100] = {0};
  fgets(nbr_img, 100, f_img); // calcul le nombre d'image à transformer
  pclose(f_img);

  // Pour la barre de chargement
  float eps = 1.5; // pourcentage équivalent à 1 '#' dans la barre
  int taille = 0;
  char barre[200] = "Traitement ascii des images";
  float max_it = atoi(nbr_img);
  init_barre_chargement(barre,&taille,eps,max_it);

  temps_kernel = (double*) malloc(max_it*sizeof(double));

  start = get_time();
  for(int k=0; k<max_it;k++){

    barre_chargement(barre,100*(k+1)/max_it,k+1,max_it,eps,taille);

    char PathName[100] = "images/frame";
    sprintf(num, "%d", k);
    strcat(PathName, num);
    strcat(PathName,".jpg");

    FreeImage_Initialise();

    // load and decode a regular file
    FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);
    FIBITMAP* bitmap = FreeImage_Load(FIF_JPEG, PathName, 0);

    if(k==0){
      declaration_1(bitmap,&width,&height,&pitch,img,argc,atoi(argv[1]),atoi(argv[2]),atoi(argv[3]),atoi(argv[4]));
      sz_in_bytes = sizeof(unsigned int) * 3 * width * height; //nb de valeurs pour toute image
      img = (unsigned int*) malloc(sz_in_bytes);
    }

  	//### allocation device ###
  	hipMalloc((void**)&d_img, sz_in_bytes);

    REORDER_IMG(img,height,width,pitch,bitmap);

    hipMemcpy(d_img, img, sz_in_bytes,hipMemcpyHostToDevice);

    declaration_2(&block_x, &block_y, &grid_x, &grid_y, width, height, atoi(argv[1]), atoi(argv[2]), k);

    //##############################################################################################
    //################################### Traitement ascii d'image #################################
    //##############################################################################################

    if(k==0){
      declaration_3(&DETAIL, &grid_x_ascii,&grid_y_ascii,&block_x_ascii,&block_y_ascii,img_ascii,width,height,atof(argv[3]),atoi(argv[4]));
      img_ascii = (float*) malloc(sizeof(float)*grid_x_ascii*grid_y_ascii);
    }

    dim3 dimBlock_ascii(block_x_ascii,block_y_ascii,1);
    dim3 dimGrid_ascii(grid_x_ascii,grid_y_ascii,1);
  
    hipMalloc((void**)&d_img_ascii, grid_x_ascii*grid_y_ascii*sizeof(float));
    start_kernel = get_time();
    Niveau_Gris_Moyennage<<<dimGrid_ascii, dimBlock_ascii>>>(d_img_ascii,d_img, width, height);
    stop_kernel = get_time();
    temps_kernel[k] = stop_kernel-start_kernel;
    hipMemcpy(img_ascii, d_img_ascii, grid_x_ascii*grid_y_ascii*sizeof(float),hipMemcpyDeviceToHost);
    hipFree(d_img);
    hipFree(d_img_ascii);

    char tab_txt[100]; // nom de l'image txt de sortie
    char tab_png[100]; // nom de l'image png de sortie
    strcat(strcpy(tab_txt, "autre/temporaire/frame"), num);
    strcat(strcpy(tab_png, "frame"), num);
    strcat(tab_txt, ".txt");
    strcat(tab_png, ".png");

    if(k==0){
      MAX = 255; //max du niveau de gris
      final_ascii = (char*) malloc(sizeof(char)*grid_x_ascii*grid_y_ascii);
    }

    tab_to_txt(final_ascii,img_ascii,tab_txt,grid_y_ascii,grid_x_ascii,block_x_ascii,block_y_ascii,MAX,DETAIL); // Creation du fichier texte contenant l'image

    txt_to_png(width,height,grid_x_ascii,tab_txt,tab_png,MAX,DETAIL); // Creation de l'image png à l'aide du fichier txt créé precedement

  }
  stop = get_time();
  cpu_time_used = stop-start;
  free(img);
  free(final_ascii);
  free(img_ascii);


  printf("\nTemps pour le traitement d'images : %f secondes\n",cpu_time_used);
  temps_kernel_moyen = 0;
  for(int k=0; k<max_it;k++){
    temps_kernel_moyen += temps_kernel[k];
  }
  printf("\nTemps moyen pour le traitement d'un kernel : %f secondes\n\n",temps_kernel_moyen/max_it);

  printf("[------------------ ASSEMBLAGE DE LA VIDEO ------------------]\n\n");
  char assemble[100] = "python3 assemble_vid.py ";
  strcat(assemble, argv[5]);
  system(assemble);

  printf("\nVideo bien assemblée.\n");

  return 0;
}

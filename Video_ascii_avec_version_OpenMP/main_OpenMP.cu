#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <sys/time.h>
#include <omp.h>
#include "FreeImage.h"
#include "bibli.cu"
#include "host.cu"
#include "device.cu"


#define WIDTH 1920
#define HEIGHT 1024
#define BPP 24 // Since we're outputting three 8 bit RGB values

using namespace std;

int main (int argc , char** argv)
{

  printf("\n[------------------ DECOUPAGE DE LA VIDEO ------------------]\n");
  char decoupe[100] = "python3 decoupe_vid.py ";
  strcat(decoupe, argv[5]);
  system(decoupe);

  printf("[------------------ TRAITEMENT ASCII DES IMAGES ------------------]\n\n");
  system("rm -r images_ascii/");
  system("mkdir images_ascii");

  FILE * f_img = popen("find images -type f | wc -l","r");
  char nbr_img[100] = {0};
  fgets(nbr_img, 100, f_img); // calcul le nombre d'image à transformer
  pclose(f_img);

  // Pour la barre de chargement
  float eps = 2.5; // pourcentage équivalent à 1 '#' dans la barre
  int taille = 0;
  char barre[200] = "Traitement ascii des images";

  int max_it = atoi(nbr_img);

  init_barre_chargement(barre,&taille,eps,max_it);

  start = get_time();

  #pragma omp parallel num_threads(atoi(argv[6])) private(num,width,height,pitch,sz_in_bytes,img,d_img,block_x,block_y,grid_x,grid_y,DETAIL,grid_x_ascii,grid_y_ascii,block_x_ascii,block_y_ascii,nb_sleep_thread_x_all,nb_sleep_thread_y_all,nb_sleep_block_x,nb_sleep_block_y,img_ascii,d_img_ascii,MAX,final_ascii)
  {
    int TN = omp_get_thread_num();
    hipStream_t num_stream_id;
    hipStreamCreate(&num_stream_id);
    int cpt_thread = 0;
    #pragma omp for
    for(int k=0; k<max_it;k++){

      if(TN==0){
        barre_chargement_openmp(barre,atoi(argv[6])*100*(k+1)/max_it,k,max_it/atoi(argv[6]),eps,taille,max_it);
      }

      char PathName[100] = "images/frame";
      sprintf(num, "%d", k);
      strcat(PathName, num);
      strcat(PathName,".jpg");

      FreeImage_Initialise();

      // load and decode a regular file
      FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);
      FIBITMAP* bitmap = FreeImage_Load(FIF_JPEG, PathName, 0);

      if(cpt_thread==0){
        declaration_1(bitmap,&width,&height,&pitch,img,argc,atoi(argv[1]),atoi(argv[2]),atoi(argv[3]),atoi(argv[4]));
        sz_in_bytes = sizeof(unsigned int) * 3 * width * height; //nb de valeurs pour toute image
        img = (unsigned int*) malloc(sz_in_bytes);
      }

    	//### allocation device ###
    	hipMalloc((void**)&d_img, sz_in_bytes);

      REORDER_IMG(img,height,width,pitch,bitmap);

      hipMemcpyAsync(d_img, img, sz_in_bytes,hipMemcpyHostToDevice,num_stream_id);
      //##############################################################################################
      //################################### Prétraitement d'image ####################################
      //##############################################################################################

      declaration_2(&block_x, &block_y, &grid_x, &grid_y, width, height, atoi(argv[1]), atoi(argv[2]), cpt_thread);

      dim3 dimBlock(block_x,block_y,1);
      dim3 dimGrid(grid_x,grid_y,1);
      Niveau_Gris<<<dimGrid, dimBlock,0,num_stream_id>>>(d_img, width, height);
      //##############################################################################################
      //#################################### Traitement ascii d'image #################################
      //##############################################################################################

      if(cpt_thread==0){
        declaration_3(&DETAIL, &grid_x_ascii,&grid_y_ascii,&block_x_ascii,&block_y_ascii,img_ascii,width,height,atof(argv[3]),atoi(argv[4]));
        img_ascii = (float*) malloc(sizeof(float)*grid_x_ascii*grid_y_ascii);
      }

      dim3 dimBlock_ascii(block_x_ascii,block_y_ascii,1);
      dim3 dimGrid_ascii(grid_x_ascii,grid_y_ascii,1);

      hipMalloc((void**)&d_img_ascii, grid_x_ascii*grid_y_ascii*sizeof(float));
      Niveau_Gris_Moyennage<<<dimGrid_ascii, dimBlock_ascii,0,num_stream_id>>>(d_img_ascii,d_img, width, height);
      hipFree(d_img);

      hipMemcpyAsync(img_ascii, d_img_ascii, grid_x_ascii*grid_y_ascii*sizeof(float),hipMemcpyDeviceToHost,num_stream_id);

      hipFree(d_img_ascii);

      char tab_txt[100]; // nom de l'image txt de sortie
      char tab_png[100]; // nom de l'image png de sortie
      strcat(strcpy(tab_txt, "autre/temporaire/frame"), num);
      strcat(strcpy(tab_png, "frame"), num);
      strcat(tab_txt, ".txt");
      strcat(tab_png, ".png");

      if(cpt_thread==0){
        MAX = 255; //max du niveau de gris
        final_ascii = (char*) malloc(sizeof(char)*grid_x_ascii*grid_y_ascii);
        cpt_thread = cpt_thread+1;
      }
      tab_to_txt(final_ascii,img_ascii,tab_txt,grid_y_ascii,grid_x_ascii,block_x_ascii,block_y_ascii,MAX,DETAIL); // Creation du fichier texte contenant l'image
      txt_to_png(width,height,grid_x_ascii,tab_txt,tab_png,MAX,DETAIL); // Creation de l'image png à l'aide du fichier txt créé precedement
      remove(tab_txt);
    }
  }
  stop = get_time();
  cpu_time_used = stop-start;
  free(img);
  free(final_ascii);
  free(img_ascii);


  printf("\nTemps pour le traitement d'images : %f secondes\n\n",cpu_time_used);

  printf("[------------------ ASSEMBLAGE DE LA VIDEO ------------------]\n\n");
  char assemble[100] = "python3 assemble_vid.py ";
  strcat(assemble, argv[5]);
  system(assemble);

  printf("\nVideo bien assemblée.\n");

  return 0;
}
